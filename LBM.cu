#include "hip/hip_runtime.h"
/*
*   The Lattice Boltzmann Method with ETHD convection
*   Yifei Guan
*   Rice University
*   Apr/12/2020
*
*/
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <hip/hip_runtime.h>
#include "LBM.h"
#include <hip/hip_runtime.h>
#define MAX(a, b) (((a) > (b)) ? (a) : (b)) 


__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y, unsigned int z)
{
    return NX*(NY*z + y)+x;
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y, unsigned int z)
{
    return NX*(NY*z + y)+x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int z, unsigned int d)
{
    return (NX*(NY*(NZ*(d-1)+z)+y)+x);
}

#define checkCudaErrors(err)  __checkCudaErrors(err,#err,__FILE__,__LINE__)
#define getLastCudaError(msg)  __getLastCudaError(msg,__FILE__,__LINE__)

inline void __checkCudaErrors(hipError_t err, const char *const func, const char *const file, const int line )
{
    if(err != hipSuccess)
    {
        fprintf(stderr, "CUDA error at %s(%d)\"%s\": [%d] %s.\n",
                file, line, func, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

inline void __getLastCudaError(const char *const errorMessage, const char *const file, const int line )
{
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s(%d): [%d] %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(-1);
    }
}

// forward declarations of kernels
__global__ void gpu_initialization(double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*);
//__global__ void gpu_taylor_green(unsigned int,double*,double*,double*);
__global__ void gpu_init_equilibrium(double*,double*,double*, double*, double*,double*,double*, double*, double*, double*, double*, double*,
	double*, double*, double*, double*, double*, double*);
__global__ void gpu_collide_save(double*,double*,double*,double*,double*,double*, double*, double*, double*, double*, double*,
	double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double,double*);
__global__ void gpu_boundary(double*, double*, double*, double*, double*, double*,double*, double*, double*, double*, double*, double*, double*);
__global__ void gpu_stream(double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*);
__global__ void gpu_bc_charge(double*, double*, double*, double*, double*, double*, double*, double*, double*);
__global__ void gpu_PBE(double*, double*, double*);
__global__ void gpu_PBE_phi(double*, double*);

__host__ void initialization(double *r, double *c, double *cn, double *fi, double *u, double *v, double *w, double *ex, double *ey, double *ez, double *temp)
{
	// blocks in grid
	dim3 grid(NX / nThreads, NY, NZ);

	// threads in block
	dim3 threads(nThreads, 1, 1);
	
	gpu_initialization << <grid, threads >> > (r, c, cn, fi, u, v, w, ex, ey, ez,temp);
	
	// Use PB equation as the charge density and electric potential initial conditions
	checkCudaErrors(hipMalloc((void**)&phi_old_gpu, mem_size_scalar));
	double *phi_old_host = (double*)malloc(mem_size_scalar);

	CHECK(hipMemcpy(phi_old_host, fi,
		mem_size_scalar, hipMemcpyDeviceToHost));

	CHECK(hipMemcpy(phi_old_gpu, phi_old_host,
		mem_size_scalar, hipMemcpyHostToDevice));


	for (unsigned int i = 0; i <= 500; ++i) {
	
		gpu_PBE << <grid, threads >> > (c, fi, cn);
	
		// =========================================================================
		// Fast poisson solver
		// =========================================================================
		fast_Poisson(charge_gpu, chargen_gpu, kx, ky, kz, plan);

		gpu_PBE_phi << <grid, threads >> > (fi, phi_old_gpu);


		CHECK(hipMemcpy(phi_old_host, fi,
			mem_size_scalar, hipMemcpyDeviceToHost));
		CHECK(hipMemcpy(phi_old_gpu, phi_old_host,
			mem_size_scalar, hipMemcpyHostToDevice));

	}
	free(phi_old_host);
	checkCudaErrors(hipFree(phi_old_gpu));
}

__global__ void gpu_initialization(double *r, double *c, double *cn, double *fi, double *u, double *v, double *w, double *ex, double *ey, double *ez,double *temp)
{
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t sidx = gpu_scalar_index(x, y, z);
	r[sidx]  = rho0;
	c[sidx]  = 0.0;
	cn[sidx] = 0.0;
	fi[sidx] = voltage;
	u[sidx]  = 0.0;
	v[sidx]  = 0.0;
	w[sidx]  = 0.0;
	ex[sidx] = 0.0;
	ey[sidx] = 0.0;
	ez[sidx] = 0.0;
	temp[sidx] = TH * (Lz - dz*z) / Lz;
}


__global__ void gpu_PBE_phi(double *fi, double *phi_old) {
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t sidx = gpu_scalar_index(x, y, z);
	fi[sidx] = PB_omega*fi[sidx] + (1.0 - PB_omega)*phi_old[sidx];
}

__global__ void gpu_PBE(double *c, double *fi, double *cn) {
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t sidx = gpu_scalar_index(x, y, z);
	c[sidx] = chargeinf*exp(-electron*fi[sidx] / kB / roomT);
	cn[sidx] = chargeinf*exp(electron*fi[sidx] / kB / roomT);
}



__host__ void init_equilibrium(double *f0, double *f1, double *h0, double *h1, double *hn0, double *hn1, double *temp0, double *temp1, double *r, double *c, double *cn,
								double *u, double *v, double *w, double *ex, double *ey, double *ez, double *temp)
{
    // blocks in grid
    dim3  grid(NX/nThreads, NY, NZ);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_init_equilibrium<<< grid, threads >>>(f0,f1,h0,h1, hn0,hn1,temp0,temp1,r,c,cn,u,v,w,ex,ey,ez,temp);
    getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f0, double *f1, double *h0, double *h1, double *hn0, double *hn1, double *temp0, double *temp1, double *r, double *c, double *cn,
										double *u, double *v, double *w, double *ex, double *ey, double *ez, double*temp)
{
    unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;
    
    double rho    = r[gpu_scalar_index(x,y,z)];
    double ux     = u[gpu_scalar_index(x,y,z)];
    double uy     = v[gpu_scalar_index(x,y,z)];
	double uz     = w[gpu_scalar_index(x,y,z)];
	double charge = c[gpu_scalar_index(x,y,z)];
	double chargen = cn[gpu_scalar_index(x, y, z)];

	double Ex     = ex[gpu_scalar_index(x,y,z)];
	double Ey     = ey[gpu_scalar_index(x,y,z)];
	double Ez     = ez[gpu_scalar_index(x,y,z)];
	double Temp   = temp[gpu_scalar_index(x,y,z)];

    // load equilibrium
    // feq_i  = w_i rho [1 + 3(ci . u) + (9/2) (ci . u)^2 - (3/2) (u.u)]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u) + (1/2) (ci . 3u)^2]
    // feq_i  = w_i rho [1 - 3/2 (u.u) + (ci . 3u){ 1 + (1/2) (ci . 3u) }]
    
    // temporary variables
    double w0r = w0*rho;
    double wsr = ws*rho;
	double war = wa*rho;
    double wdr = wd*rho;

	double w0c = w0*charge;
	double wsc = ws*charge;
	double wac = wa*charge;
	double wdc = wd*charge;

	double w0cn = w0*chargen;
	double wscn = ws*chargen;
	double wacn = wa*chargen;
	double wdcn = wd*chargen;

	double w0t = w0*Temp;
	double wst = ws*Temp;
	double wat = wa*Temp;
	double wdt = wd*Temp;

    double omusq   = 1.0 - 0.5*(ux*ux+uy*uy+uz*uz)/cs_square;
	double omusq_c = 1.0 - 0.5*((ux + K*Ex)*(ux + K*Ex) + (uy + K*Ey)*(uy + K*Ey) + (uz + K*Ez)*(uz + K*Ez)) / cs_square;
	double omusq_cn = 1.0 - 0.5*((ux + Kn*Ex)*(ux + Kn*Ex) + (uy + Kn*Ey)*(uy + Kn*Ey) + (uz + Kn*Ez)*(uz + Kn*Ez)) / cs_square;

    
    double tux   = ux / cs_square / CFL;
    double tuy   = uy / cs_square / CFL;
	double tuz   = uz / cs_square / CFL;
	double tux_c = (ux + K*Ex) / cs_square / CFL;
	double tuy_c = (uy + K*Ey) / cs_square / CFL;
	double tuz_c = (uz + K*Ez) / cs_square / CFL;
	double tux_cn = (ux + Kn*Ex) / cs_square / CFL;
	double tuy_cn = (uy + Kn*Ey) / cs_square / CFL;
	double tuz_cn = (uz + Kn*Ez) / cs_square / CFL;
    
	// zero weight
    f0[gpu_field0_index(x,y,z)]      = w0r*(omusq);
	h0[gpu_field0_index(x,y,z)]      = w0c*(omusq_c);
	hn0[gpu_field0_index(x, y, z)]   = w0cn*(omusq_cn);
	temp0[gpu_field0_index(x, y, z)] = w0t*(omusq);
    
	// adjacent weight
	// flow
    double cidot3u = tux;
    f1[gpu_fieldn_index(x,y,z,1)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
	cidot3u = -tux;
	f1[gpu_fieldn_index(x,y,z,2)]  = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
    cidot3u = tuy;
    f1[gpu_fieldn_index(x,y,z,3)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tuy;
    f1[gpu_fieldn_index(x,y,z,4)]  = wsr*(omusq + cidot3u*(1.0+0.5*cidot3u));
	cidot3u = tuz;
	f1[gpu_fieldn_index(x,y,z,5)] = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz;
	f1[gpu_fieldn_index(x,y,z,6)] = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// charge
	cidot3u = tux_c;
	h1[gpu_fieldn_index(x,y,z,1)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c;
	h1[gpu_fieldn_index(x,y,z,2)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c;
	h1[gpu_fieldn_index(x,y,z,3)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c;
	h1[gpu_fieldn_index(x,y,z,4)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c;
	h1[gpu_fieldn_index(x,y,z,5)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz_c;
	h1[gpu_fieldn_index(x,y,z,6)] = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 1)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 2)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 3)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 4)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 5)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 6)] = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	// temperature
	cidot3u = tux;
	temp1[gpu_fieldn_index(x, y, z, 1)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux;
	temp1[gpu_fieldn_index(x, y, z, 2)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy;
	temp1[gpu_fieldn_index(x, y, z, 3)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy;
	temp1[gpu_fieldn_index(x, y, z, 4)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz;
	temp1[gpu_fieldn_index(x, y, z, 5)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz;
	temp1[gpu_fieldn_index(x, y, z, 6)] = wst*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// diagonal weight
	// flow
    cidot3u = tux+tuy;
    f1[gpu_fieldn_index(x,y,z,7)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tuy-tux;
    f1[gpu_fieldn_index(x,y,z,8)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = tux+tuz;
    f1[gpu_fieldn_index(x,y,z,9)]  = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
    cidot3u = -tux-tuz;
    f1[gpu_fieldn_index(x,y,z,10)] = war*(omusq + cidot3u*(1.0+0.5*cidot3u));
	cidot3u = tuz + tuy;
	f1[gpu_fieldn_index(x,y,z,11)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tuz;
	f1[gpu_fieldn_index(x,y,z,12)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy;
	f1[gpu_fieldn_index(x,y,z,13)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux;
	f1[gpu_fieldn_index(x,y,z,14)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuz;
	f1[gpu_fieldn_index(x,y,z,15)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux;
	f1[gpu_fieldn_index(x,y,z,16)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tuz;
	f1[gpu_fieldn_index(x,y,z,17)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tuy;
	f1[gpu_fieldn_index(x,y,z,18)] = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// charge
	cidot3u = tux_c + tuy_c;
	h1[gpu_fieldn_index(x, y, z, 7)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c - tux_c;
	h1[gpu_fieldn_index(x, y, z, 8)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuz_c;
	h1[gpu_fieldn_index(x, y, z, 9)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 10)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c + tuz_c;
	h1[gpu_fieldn_index(x, y, z, 11)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 12)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuy_c;
	h1[gpu_fieldn_index(x, y, z, 13)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tux_c;
	h1[gpu_fieldn_index(x, y, z, 14)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 15)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tux_c;
	h1[gpu_fieldn_index(x, y, z, 16)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 17)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tuy_c;
	h1[gpu_fieldn_index(x, y, z, 18)] = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn + tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 7)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn - tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 8)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 9)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 10)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn + tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 11)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 12)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 13)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 14)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 15)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 16)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 17)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 18)] = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	// temperature
	cidot3u = tux + tuy;
	temp1[gpu_fieldn_index(x, y, z, 7)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tux;
	temp1[gpu_fieldn_index(x, y, z, 8)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz;
	temp1[gpu_fieldn_index(x, y, z, 9)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux - tuz;
	temp1[gpu_fieldn_index(x, y, z, 10)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz;
	temp1[gpu_fieldn_index(x, y, z, 11)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tuz;
	temp1[gpu_fieldn_index(x, y, z, 12)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy;
	temp1[gpu_fieldn_index(x, y, z, 13)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux;
	temp1[gpu_fieldn_index(x, y, z, 14)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuz;
	temp1[gpu_fieldn_index(x, y, z, 15)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux;
	temp1[gpu_fieldn_index(x, y, z, 16)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tuz;
	temp1[gpu_fieldn_index(x, y, z, 17)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tuy;
	temp1[gpu_fieldn_index(x, y, z, 18)] = wat*(omusq + cidot3u*(1.0 + 0.5*cidot3u));


	// 3d diagonal
	//flow
	cidot3u = tux + tuy + tuz;
	f1[gpu_fieldn_index(x, y, z, 19)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tux - tuz;
	f1[gpu_fieldn_index(x, y, z, 20)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuy - tuz;
	f1[gpu_fieldn_index(x, y, z, 21)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux - tuy;
	f1[gpu_fieldn_index(x, y, z, 22)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz - tuy;
	f1[gpu_fieldn_index(x, y, z, 23)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux - tuz;
	f1[gpu_fieldn_index(x, y, z, 24)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz - tux;
	f1[gpu_fieldn_index(x, y, z, 25)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy - tuz;
	f1[gpu_fieldn_index(x, y, z, 26)] = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	//charge
	cidot3u = tux_c + tuy_c + tuz_c;
	h1[gpu_fieldn_index(x, y, z, 19)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c -tuy_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 20)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuy_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 21)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tux_c - tuy_c;
	h1[gpu_fieldn_index(x, y, z, 22)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuz_c - tuy_c;
	h1[gpu_fieldn_index(x, y, z, 23)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tux_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 24)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c + tuz_c - tux_c;
	h1[gpu_fieldn_index(x, y, z, 25)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuy_c - tuz_c;
	h1[gpu_fieldn_index(x, y, z, 26)] = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn + tuy_cn + tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 19)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn - tuy_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 20)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuy_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 21)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tux_cn - tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 22)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuz_cn - tuy_cn;
	hn1[gpu_fieldn_index(x, y, z, 23)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tux_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 24)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn + tuz_cn - tux_cn;
	hn1[gpu_fieldn_index(x, y, z, 25)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuy_cn - tuz_cn;
	hn1[gpu_fieldn_index(x, y, z, 26)] = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	//temperature
	cidot3u = tux + tuy + tuz;
	temp1[gpu_fieldn_index(x, y, z, 19)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux - tuy - tuz;
	temp1[gpu_fieldn_index(x, y, z, 20)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuy - tuz;
	temp1[gpu_fieldn_index(x, y, z, 21)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux - tuy;
	temp1[gpu_fieldn_index(x, y, z, 22)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz - tuy;
	temp1[gpu_fieldn_index(x, y, z, 23)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux - tuz;
	temp1[gpu_fieldn_index(x, y, z, 24)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz - tux;
	temp1[gpu_fieldn_index(x, y, z, 25)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy - tuz;
	temp1[gpu_fieldn_index(x, y, z, 26)] = wdt*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
}

__host__ void stream_collide_save(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *hn0, double *hn1, double *hn2, 
	double *temp0, double *temp1, double *temp2,
	double *r, double *c, double *cn, double *u, double *v, double *w, double *ex, double *ey, double *ez, double *Temp, double t,double *f0bc)
{
    // blocks in grid
    dim3  grid(NX/nThreads, NY, NZ);
    // threads in block
    dim3  threads(nThreads, 1, 1);

    gpu_collide_save<<< grid, threads >>>(f0,f1,f2, h0, h1, h2, hn0, hn1, hn2, temp0, temp1, temp2, r, c, cn,u,v,w, ex, ey,ez,Temp,t,f0bc);
	gpu_boundary << < grid, threads >> >(f0, f1, f2, h0, h1, h2, hn0, hn1, hn2, temp0, temp1, temp2, f0bc);
	gpu_stream << < grid, threads >> >(f0, f1, f2, h0, h1, h2, hn0, hn1, hn2, temp0, temp1, temp2);
	gpu_bc_charge << < grid, threads >> >(h0, h1, h2, hn0, hn1, hn2, temp0, temp1, temp2);


    getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_collide_save(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *hn0, double *hn1, double *hn2, 
	double *temp0, double *temp1, double *temp2,
	double *r, double *c, double *cn, double *u, double *v, double *w, double *ex, double *ey, double *ez, double *Temperature, double t,double *f0bc)
{
	// useful constants
	double omega_plus = 1.0 / (nu / cs_square / dt + 1.0 / 2.0) / dt;
	double omega_minus = 1.0 / (V / (nu / cs_square / dt) + 1.0 / 2.0) / dt;
	double omega_c_minus = 1.0 / (diffu / cs_square / dt + 1.0 / 2.0) / dt;
	double omega_c_plus = 1.0 / (VC / (diffu / cs_square / dt) + 1.0 / 2.0) / dt;
	double omega_cn_minus = 1.0 / (diffun / cs_square / dt + 1.0 / 2.0) / dt;
	double omega_cn_plus = 1.0 / (VCn / (diffun / cs_square / dt) + 1.0 / 2.0) / dt;
	double omega_T_minus = 1.0 / (D / cs_square / dt + 1.0 / 2.0) / dt;
	double omega_T_plus = 1.0 / (VT / (D / cs_square / dt) + 1.0 / 2.0) / dt;

	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	// storage of f0 at upper and lower plate
	if (z == 0)  f0bc[gpu_field0_index(x, y, 0)] = f0[gpu_field0_index(x, y, z)]; // lower plate
	
	if (z==NZ-1) f0bc[gpu_field0_index(x, y, 1)] = f0[gpu_field0_index(x, y, z)]; // upper plate

	// load populations from nodes (ft is the same as f1)
	double ft0    = f0[gpu_field0_index(x, y, z)];
	double ht0    = h0[gpu_field0_index(x, y, z)];
	double hnt0	  = hn0[gpu_field0_index(x, y, z)];
	double tempt0 = temp0[gpu_field0_index(x, y, z)];

	double ft1  = f1[gpu_fieldn_index(x, y, z, 1)];
	double ft2  = f1[gpu_fieldn_index(x, y, z, 2)];
	double ft3  = f1[gpu_fieldn_index(x, y, z, 3)];
	double ft4  = f1[gpu_fieldn_index(x, y, z, 4)];
	double ft5  = f1[gpu_fieldn_index(x, y, z, 5)];
	double ft6  = f1[gpu_fieldn_index(x, y, z, 6)];
	double ft7  = f1[gpu_fieldn_index(x, y, z, 7)];
	double ft8  = f1[gpu_fieldn_index(x, y, z, 8)];
	double ft9  = f1[gpu_fieldn_index(x, y, z, 9)];
	double ft10 = f1[gpu_fieldn_index(x, y, z, 10)];
	double ft11 = f1[gpu_fieldn_index(x, y, z, 11)];
	double ft12 = f1[gpu_fieldn_index(x, y, z, 12)];
	double ft13 = f1[gpu_fieldn_index(x, y, z, 13)];
	double ft14 = f1[gpu_fieldn_index(x, y, z, 14)];
	double ft15 = f1[gpu_fieldn_index(x, y, z, 15)];
	double ft16 = f1[gpu_fieldn_index(x, y, z, 16)];
	double ft17 = f1[gpu_fieldn_index(x, y, z, 17)];
	double ft18 = f1[gpu_fieldn_index(x, y, z, 18)];
	double ft19 = f1[gpu_fieldn_index(x, y, z, 19)];
	double ft20 = f1[gpu_fieldn_index(x, y, z, 20)];
	double ft21 = f1[gpu_fieldn_index(x, y, z, 21)];
	double ft22 = f1[gpu_fieldn_index(x, y, z, 22)];
	double ft23 = f1[gpu_fieldn_index(x, y, z, 23)];
	double ft24 = f1[gpu_fieldn_index(x, y, z, 24)];
	double ft25 = f1[gpu_fieldn_index(x, y, z, 25)];
	double ft26 = f1[gpu_fieldn_index(x, y, z, 26)];
	
	double ht1  = h1[gpu_fieldn_index(x, y, z, 1)];
	double ht2  = h1[gpu_fieldn_index(x, y, z, 2)];
	double ht3  = h1[gpu_fieldn_index(x, y, z, 3)];
	double ht4  = h1[gpu_fieldn_index(x, y, z, 4)];
	double ht5  = h1[gpu_fieldn_index(x, y, z, 5)];
	double ht6  = h1[gpu_fieldn_index(x, y, z, 6)];
	double ht7  = h1[gpu_fieldn_index(x, y, z, 7)];
	double ht8  = h1[gpu_fieldn_index(x, y, z, 8)];
	double ht9  = h1[gpu_fieldn_index(x, y, z, 9)];
	double ht10 = h1[gpu_fieldn_index(x, y, z, 10)];
	double ht11 = h1[gpu_fieldn_index(x, y, z, 11)];
	double ht12 = h1[gpu_fieldn_index(x, y, z, 12)];
	double ht13 = h1[gpu_fieldn_index(x, y, z, 13)];
	double ht14 = h1[gpu_fieldn_index(x, y, z, 14)];
	double ht15 = h1[gpu_fieldn_index(x, y, z, 15)];
	double ht16 = h1[gpu_fieldn_index(x, y, z, 16)];
	double ht17 = h1[gpu_fieldn_index(x, y, z, 17)];
	double ht18 = h1[gpu_fieldn_index(x, y, z, 18)];
	double ht19 = h1[gpu_fieldn_index(x, y, z, 19)];
	double ht20 = h1[gpu_fieldn_index(x, y, z, 20)];
	double ht21 = h1[gpu_fieldn_index(x, y, z, 21)];
	double ht22 = h1[gpu_fieldn_index(x, y, z, 22)];
	double ht23 = h1[gpu_fieldn_index(x, y, z, 23)];
	double ht24 = h1[gpu_fieldn_index(x, y, z, 24)];
	double ht25 = h1[gpu_fieldn_index(x, y, z, 25)];
	double ht26 = h1[gpu_fieldn_index(x, y, z, 26)];

	double hnt1 = hn1[gpu_fieldn_index(x, y, z, 1)];
	double hnt2 = hn1[gpu_fieldn_index(x, y, z, 2)];
	double hnt3 = hn1[gpu_fieldn_index(x, y, z, 3)];
	double hnt4 = hn1[gpu_fieldn_index(x, y, z, 4)];
	double hnt5 = hn1[gpu_fieldn_index(x, y, z, 5)];
	double hnt6 = hn1[gpu_fieldn_index(x, y, z, 6)];
	double hnt7 = hn1[gpu_fieldn_index(x, y, z, 7)];
	double hnt8 = hn1[gpu_fieldn_index(x, y, z, 8)];
	double hnt9 = hn1[gpu_fieldn_index(x, y, z, 9)];
	double hnt10 = hn1[gpu_fieldn_index(x, y, z, 10)];
	double hnt11 = hn1[gpu_fieldn_index(x, y, z, 11)];
	double hnt12 = hn1[gpu_fieldn_index(x, y, z, 12)];
	double hnt13 = hn1[gpu_fieldn_index(x, y, z, 13)];
	double hnt14 = hn1[gpu_fieldn_index(x, y, z, 14)];
	double hnt15 = hn1[gpu_fieldn_index(x, y, z, 15)];
	double hnt16 = hn1[gpu_fieldn_index(x, y, z, 16)];
	double hnt17 = hn1[gpu_fieldn_index(x, y, z, 17)];
	double hnt18 = hn1[gpu_fieldn_index(x, y, z, 18)];
	double hnt19 = hn1[gpu_fieldn_index(x, y, z, 19)];
	double hnt20 = hn1[gpu_fieldn_index(x, y, z, 20)];
	double hnt21 = hn1[gpu_fieldn_index(x, y, z, 21)];
	double hnt22 = hn1[gpu_fieldn_index(x, y, z, 22)];
	double hnt23 = hn1[gpu_fieldn_index(x, y, z, 23)];
	double hnt24 = hn1[gpu_fieldn_index(x, y, z, 24)];
	double hnt25 = hn1[gpu_fieldn_index(x, y, z, 25)];
	double hnt26 = hn1[gpu_fieldn_index(x, y, z, 26)];

	double tempt1 = temp1[gpu_fieldn_index(x, y, z, 1)];
	double tempt2 = temp1[gpu_fieldn_index(x, y, z, 2)];
	double tempt3 = temp1[gpu_fieldn_index(x, y, z, 3)];
	double tempt4 = temp1[gpu_fieldn_index(x, y, z, 4)];
	double tempt5 = temp1[gpu_fieldn_index(x, y, z, 5)];
	double tempt6 = temp1[gpu_fieldn_index(x, y, z, 6)];
	double tempt7 = temp1[gpu_fieldn_index(x, y, z, 7)];
	double tempt8 = temp1[gpu_fieldn_index(x, y, z, 8)];
	double tempt9 = temp1[gpu_fieldn_index(x, y, z, 9)];
	double tempt10 = temp1[gpu_fieldn_index(x, y, z, 10)];
	double tempt11 = temp1[gpu_fieldn_index(x, y, z, 11)];
	double tempt12 = temp1[gpu_fieldn_index(x, y, z, 12)];
	double tempt13 = temp1[gpu_fieldn_index(x, y, z, 13)];
	double tempt14 = temp1[gpu_fieldn_index(x, y, z, 14)];
	double tempt15 = temp1[gpu_fieldn_index(x, y, z, 15)];
	double tempt16 = temp1[gpu_fieldn_index(x, y, z, 16)];
	double tempt17 = temp1[gpu_fieldn_index(x, y, z, 17)];
	double tempt18 = temp1[gpu_fieldn_index(x, y, z, 18)];
	double tempt19 = temp1[gpu_fieldn_index(x, y, z, 19)];
	double tempt20 = temp1[gpu_fieldn_index(x, y, z, 20)];
	double tempt21 = temp1[gpu_fieldn_index(x, y, z, 21)];
	double tempt22 = temp1[gpu_fieldn_index(x, y, z, 22)];
	double tempt23 = temp1[gpu_fieldn_index(x, y, z, 23)];
	double tempt24 = temp1[gpu_fieldn_index(x, y, z, 24)];
	double tempt25 = temp1[gpu_fieldn_index(x, y, z, 25)];
	double tempt26 = temp1[gpu_fieldn_index(x, y, z, 26)];

	// compute macroscopic variables from microscopic variables
	double rho = ft0 + ft1 + ft2 + ft3 + ft4 + ft5 + ft6 + ft7 + ft8 
		+ ft9 + ft10 + ft11 + ft12 + ft13 + ft14 + ft15 + ft16 + ft17 + ft18 + ft19 + ft20 + ft21 + ft22 + ft23 + ft24 + ft25 + ft26;
	double rhoinv = 1.0 / rho;
	double charge = ht0 + ht1 + ht2 + ht3 + ht4 + ht5 + ht6 + ht7 + ht8 + ht9
		+ ht10 + ht11 + ht12 + ht13 + ht14 + ht15 + ht16 + ht17 + ht18 + ht19 + ht20 + ht21 + ht22 + ht23 + ht24 + ht25 + ht26;
	double chargen = hnt0 + hnt1 + hnt2 + hnt3 + hnt4 + hnt5 + hnt6 + hnt7 + hnt8 + hnt9
		+ hnt10 + hnt11 + hnt12 + hnt13 + hnt14 + hnt15 + hnt16 + hnt17 + hnt18 + hnt19 + hnt20 + hnt21 + hnt22 + hnt23 + hnt24 + hnt25 + hnt26;
	double temp = tempt0 + tempt1 + tempt2 + tempt3 + tempt4 + tempt5 + tempt6 + tempt7 + tempt8 + tempt9
		+ tempt10 + tempt11 + tempt12 + tempt13 + tempt14 + tempt15 + tempt16 + tempt17 + tempt18 + tempt19 + tempt20 + tempt21
		+ tempt22 + tempt23 + tempt24 + tempt25 + tempt26;

	double Ex = ex[gpu_scalar_index(x, y, z)];
	double Ey = ey[gpu_scalar_index(x, y, z)];
	double Ez = ez[gpu_scalar_index(x, y, z)];
	double forcex = convertCtoCharge*(charge-chargen) * (Ex+Ext) + exf;
	double forcey = convertCtoCharge*(charge - chargen) * Ey;
	double forcez = convertCtoCharge*(charge - chargen) * Ez + rho0*temp*Ra*nu*D;

	double ux = rhoinv*((ft1 + ft7 + ft9  + ft13 + ft15 + ft19 + ft21 + ft23 + ft26 
						- (ft2 + ft8 + ft10 + ft14 + ft16 + ft20 + ft22 + ft24 + ft25)) / CFL + forcex*dt*0.5);
	double uy = rhoinv*((ft3 + ft7 + ft11 + ft14 + ft17 + ft19 + ft21 + ft24 + ft25
						- (ft4 + ft8 + ft12 + ft13 + ft18 + ft20 + ft22 + ft23 + ft26)) / CFL + forcey*dt*0.5);
	double uz = rhoinv*((ft5 + ft9 + ft11 + ft16 + ft18 + ft19 + ft22 + ft23 + ft25
						- (ft6 + ft10 + ft12 + ft15 + ft17 + ft20 + ft21 + ft24 + ft26)) / CFL + forcez*dt*0.5);
	
	if (perturb==1){
		double xx = x*dx;
		double yy = y*dy;
		double zz = (z-0.5)*dz;
		// Square patterns
		uz = (cos(2 * M_PI*zz) - 1)*cos(2 * M_PI / LL*xx)*cos(2 * M_PI / LL*yy);
		ux = 0.5 * LL*sin(2 * M_PI*zz)*sin(2 * M_PI / LL*xx)*cos(2 * M_PI / LL*yy);
		uy = 0.5 * LL*sin(2 * M_PI*zz)*sin(2 * M_PI / LL*yy)*cos(2 * M_PI / LL*xx);
		// Hexagon patterns
		/*	double L = 0.5; // ratio of wavelength to domain size
		double a = 4 * M_PI / 3 / L;
		uz = (cos(2 * M_PI*z) - 1) / 3 *(2 * cos(2 * M_PI / (sqrtf(3)*L)*x)*cos(2 * M_PI / (3 * L)*y) + cos(4 * M_PI / (3 * L)*y));
		ux = 2 * M_PI*sin(2 * M_PI*z)*(4 * M_PI) / (3 * sqrtf(3)*L*a ^ 2)*sin(2 * M_PI / (sqrtf(3)*L)*x)*cos(2 * M_PI / (3 * L)*y);
		uy = 2 * M_PI*sin(2 * M_PI*z)*(4 * M_PI) / (9 * L*a ^ 2)*(cos(2 * M_PI / (sqrtf(3)*L)*x) + 2 * cos(2 * M_PI / (3 * L)*y))*sin(2 * M_PI / (3 * L)*y);
		//charge = charge + uz;*/
	}
	else{
		if (z == 0) {
			double ftm0 = f0[gpu_field0_index(x, y, 1)];
			double htm0 = h0[gpu_field0_index(x, y, 1)];
			double hntm0 = hn0[gpu_field0_index(x, y, 1)];
			double temptm0 = temp0[gpu_field0_index(x, y, 1)];

			double ftm1 = f1[gpu_fieldn_index(x, y, 1, 1)];
			double ftm2 = f1[gpu_fieldn_index(x, y, 1, 2)];
			double ftm3 = f1[gpu_fieldn_index(x, y, 1, 3)];
			double ftm4 = f1[gpu_fieldn_index(x, y, 1, 4)];
			double ftm5 = f1[gpu_fieldn_index(x, y, 1, 5)];
			double ftm6 = f1[gpu_fieldn_index(x, y, 1, 6)];
			double ftm7 = f1[gpu_fieldn_index(x, y, 1, 7)];
			double ftm8 = f1[gpu_fieldn_index(x, y, 1, 8)];
			double ftm9 = f1[gpu_fieldn_index(x, y, 1, 9)];
			double ftm10 = f1[gpu_fieldn_index(x, y, 1, 10)];
			double ftm11 = f1[gpu_fieldn_index(x, y, 1, 11)];
			double ftm12 = f1[gpu_fieldn_index(x, y, 1, 12)];
			double ftm13 = f1[gpu_fieldn_index(x, y, 1, 13)];
			double ftm14 = f1[gpu_fieldn_index(x, y, 1, 14)];
			double ftm15 = f1[gpu_fieldn_index(x, y, 1, 15)];
			double ftm16 = f1[gpu_fieldn_index(x, y, 1, 16)];
			double ftm17 = f1[gpu_fieldn_index(x, y, 1, 17)];
			double ftm18 = f1[gpu_fieldn_index(x, y, 1, 18)];
			double ftm19 = f1[gpu_fieldn_index(x, y, 1, 19)];
			double ftm20 = f1[gpu_fieldn_index(x, y, 1, 20)];
			double ftm21 = f1[gpu_fieldn_index(x, y, 1, 21)];
			double ftm22 = f1[gpu_fieldn_index(x, y, 1, 22)];
			double ftm23 = f1[gpu_fieldn_index(x, y, 1, 23)];
			double ftm24 = f1[gpu_fieldn_index(x, y, 1, 24)];
			double ftm25 = f1[gpu_fieldn_index(x, y, 1, 25)];
			double ftm26 = f1[gpu_fieldn_index(x, y, 1, 26)];

			double htm1 = h1[gpu_fieldn_index(x, y, 1, 1)];
			double htm2 = h1[gpu_fieldn_index(x, y, 1, 2)];
			double htm3 = h1[gpu_fieldn_index(x, y, 1, 3)];
			double htm4 = h1[gpu_fieldn_index(x, y, 1, 4)];
			double htm5 = h1[gpu_fieldn_index(x, y, 1, 5)];
			double htm6 = h1[gpu_fieldn_index(x, y, 1, 6)];
			double htm7 = h1[gpu_fieldn_index(x, y, 1, 7)];
			double htm8 = h1[gpu_fieldn_index(x, y, 1, 8)];
			double htm9 = h1[gpu_fieldn_index(x, y, 1, 9)];
			double htm10 = h1[gpu_fieldn_index(x, y, 1, 10)];
			double htm11 = h1[gpu_fieldn_index(x, y, 1, 11)];
			double htm12 = h1[gpu_fieldn_index(x, y, 1, 12)];
			double htm13 = h1[gpu_fieldn_index(x, y, 1, 13)];
			double htm14 = h1[gpu_fieldn_index(x, y, 1, 14)];
			double htm15 = h1[gpu_fieldn_index(x, y, 1, 15)];
			double htm16 = h1[gpu_fieldn_index(x, y, 1, 16)];
			double htm17 = h1[gpu_fieldn_index(x, y, 1, 17)];
			double htm18 = h1[gpu_fieldn_index(x, y, 1, 18)];
			double htm19 = h1[gpu_fieldn_index(x, y, 1, 19)];
			double htm20 = h1[gpu_fieldn_index(x, y, 1, 20)];
			double htm21 = h1[gpu_fieldn_index(x, y, 1, 21)];
			double htm22 = h1[gpu_fieldn_index(x, y, 1, 22)];
			double htm23 = h1[gpu_fieldn_index(x, y, 1, 23)];
			double htm24 = h1[gpu_fieldn_index(x, y, 1, 24)];
			double htm25 = h1[gpu_fieldn_index(x, y, 1, 25)];
			double htm26 = h1[gpu_fieldn_index(x, y, 1, 26)];

			double hntm1 = hn1[gpu_fieldn_index(x, y, 1, 1)];
			double hntm2 = hn1[gpu_fieldn_index(x, y, 1, 2)];
			double hntm3 = hn1[gpu_fieldn_index(x, y, 1, 3)];
			double hntm4 = hn1[gpu_fieldn_index(x, y, 1, 4)];
			double hntm5 = hn1[gpu_fieldn_index(x, y, 1, 5)];
			double hntm6 = hn1[gpu_fieldn_index(x, y, 1, 6)];
			double hntm7 = hn1[gpu_fieldn_index(x, y, 1, 7)];
			double hntm8 = hn1[gpu_fieldn_index(x, y, 1, 8)];
			double hntm9 = hn1[gpu_fieldn_index(x, y, 1, 9)];
			double hntm10 = hn1[gpu_fieldn_index(x, y, 1, 10)];
			double hntm11 = hn1[gpu_fieldn_index(x, y, 1, 11)];
			double hntm12 = hn1[gpu_fieldn_index(x, y, 1, 12)];
			double hntm13 = hn1[gpu_fieldn_index(x, y, 1, 13)];
			double hntm14 = hn1[gpu_fieldn_index(x, y, 1, 14)];
			double hntm15 = hn1[gpu_fieldn_index(x, y, 1, 15)];
			double hntm16 = hn1[gpu_fieldn_index(x, y, 1, 16)];
			double hntm17 = hn1[gpu_fieldn_index(x, y, 1, 17)];
			double hntm18 = hn1[gpu_fieldn_index(x, y, 1, 18)];
			double hntm19 = hn1[gpu_fieldn_index(x, y, 1, 19)];
			double hntm20 = hn1[gpu_fieldn_index(x, y, 1, 20)];
			double hntm21 = hn1[gpu_fieldn_index(x, y, 1, 21)];
			double hntm22 = hn1[gpu_fieldn_index(x, y, 1, 22)];
			double hntm23 = hn1[gpu_fieldn_index(x, y, 1, 23)];
			double hntm24 = hn1[gpu_fieldn_index(x, y, 1, 24)];
			double hntm25 = hn1[gpu_fieldn_index(x, y, 1, 25)];
			double hntm26 = hn1[gpu_fieldn_index(x, y, 1, 26)];

			double temptm1 = temp1[gpu_fieldn_index(x, y, 1, 1)];
			double temptm2 = temp1[gpu_fieldn_index(x, y, 1, 2)];
			double temptm3 = temp1[gpu_fieldn_index(x, y, 1, 3)];
			double temptm4 = temp1[gpu_fieldn_index(x, y, 1, 4)];
			double temptm5 = temp1[gpu_fieldn_index(x, y, 1, 5)];
			double temptm6 = temp1[gpu_fieldn_index(x, y, 1, 6)];
			double temptm7 = temp1[gpu_fieldn_index(x, y, 1, 7)];
			double temptm8 = temp1[gpu_fieldn_index(x, y, 1, 8)];
			double temptm9 = temp1[gpu_fieldn_index(x, y, 1, 9)];
			double temptm10 = temp1[gpu_fieldn_index(x, y, 1, 10)];
			double temptm11 = temp1[gpu_fieldn_index(x, y, 1, 11)];
			double temptm12 = temp1[gpu_fieldn_index(x, y, 1, 12)];
			double temptm13 = temp1[gpu_fieldn_index(x, y, 1, 13)];
			double temptm14 = temp1[gpu_fieldn_index(x, y, 1, 14)];
			double temptm15 = temp1[gpu_fieldn_index(x, y, 1, 15)];
			double temptm16 = temp1[gpu_fieldn_index(x, y, 1, 16)];
			double temptm17 = temp1[gpu_fieldn_index(x, y, 1, 17)];
			double temptm18 = temp1[gpu_fieldn_index(x, y, 1, 18)];
			double temptm19 = temp1[gpu_fieldn_index(x, y, 1, 19)];
			double temptm20 = temp1[gpu_fieldn_index(x, y, 1, 20)];
			double temptm21 = temp1[gpu_fieldn_index(x, y, 1, 21)];
			double temptm22 = temp1[gpu_fieldn_index(x, y, 1, 22)];
			double temptm23 = temp1[gpu_fieldn_index(x, y, 1, 23)];
			double temptm24 = temp1[gpu_fieldn_index(x, y, 1, 24)];
			double temptm25 = temp1[gpu_fieldn_index(x, y, 1, 25)];
			double temptm26 = temp1[gpu_fieldn_index(x, y, 1, 26)];

			// compute macroscopic variables from microscopic variables
			double rhom = ftm0 + ftm1 + ftm2 + ftm3 + ftm4 + ftm5 + ftm6 + ftm7 + ftm8
				+ ftm9 + ftm10 + ftm11 + ftm12 + ftm13 + ftm14 + ftm15 + ftm16 + ftm17 + ftm18 + ftm19 + ftm20 + ftm21 + ftm22 + ftm23 + ftm24 + ftm25 + ftm26;
			double rhoinvm = 1.0 / rho;
			double chargem = htm0 + htm1 + htm2 + htm3 + htm4 + htm5 + htm6 + htm7 + htm8 + htm9
				+ htm10 + htm11 + htm12 + htm13 + htm14 + htm15 + htm16 + htm17 + htm18 + htm19 + htm20 + htm21 + htm22 + htm23 + htm24 + htm25 + htm26;
			double chargenm = hntm0 + hntm1 + hntm2 + hntm3 + hntm4 + hntm5 + hntm6 + hntm7 + hntm8 + hntm9
				+ hntm10 + hntm11 + hntm12 + hntm13 + hntm14 + hntm15 + hntm16 + hntm17 + hntm18 + hntm19 + hntm20 + hntm21 + hntm22 + hntm23 + hntm24 + hntm25 + hntm26;
			double tempm = temptm0 + temptm1 + temptm2 + temptm3 + temptm4 + temptm5 + temptm6 + temptm7 + temptm8 + temptm9
				+ temptm10 + temptm11 + temptm12 + temptm13 + temptm14 + temptm15 + temptm16 + temptm17 + temptm18 + temptm19
				+ temptm20 + temptm21 + temptm22 + temptm23 + temptm24 + temptm25 + temptm26;

			double Exm = ex[gpu_scalar_index(x, y, 1)];
			double Eym = ey[gpu_scalar_index(x, y, 1)];
			double Ezm = ez[gpu_scalar_index(x, y, 1)];
			double forcexm = convertCtoCharge*(chargem - chargenm) * (Exm+Ext) + exf;
			double forceym = convertCtoCharge*(chargem - chargenm) * Eym;
			double forcezm = convertCtoCharge*(chargem - chargenm) * Ezm + rho0*tempm*Ra*nu*D;
			ux = -rhoinvm*((ftm1 + ftm7 + ftm9 + ftm13 + ftm15 + ftm19 + ftm21 + ftm23 + ftm26
				- (ftm2 + ftm8 + ftm10 + ftm14 + ftm16 + ftm20 + ftm22 + ftm24 + ftm25)) / CFL + forcexm*dt*0.5);
			uy = -rhoinvm*((ftm3 + ftm7 + ftm11 + ftm14 + ftm17 + ftm19 + ftm21 + ftm24 + ftm25
				- (ftm4 + ftm8 + ftm12 + ftm13 + ftm18 + ftm20 + ftm22 + ftm23 + ftm26)) / CFL + forceym*dt*0.5);
			uz = -rhoinvm*((ftm5 + ftm9 + ftm11 + ftm16 + ftm18 + ftm19 + ftm22 + ftm23 + ftm25
				- (ftm6 + ftm10 + ftm12 + ftm15 + ftm17 + ftm20 + ftm21 + ftm24 + ftm26)) / CFL + forcezm*dt*0.5);
		}
	}
	

	// write to memory (only when visualizing the data)
	
	r[gpu_scalar_index(x, y, z)] = rho;
	u[gpu_scalar_index(x, y, z)] = ux;
	v[gpu_scalar_index(x, y, z)] = uy;
	w[gpu_scalar_index(x, y, z)] = uz;
	c[gpu_scalar_index(x, y, z)] = charge;
	cn[gpu_scalar_index(x, y, z)] = chargen;
	Temperature[gpu_scalar_index(x, y, z)] = temp;

	// collision step
	// now compute and relax to equilibrium
	// note that
	// feq_i  = w_i rho [1 + (ci . u / cs_square) + (1/2) (ci . u / cs_square)^2 - (1/2) (u.u) / cs_square]
	// feq_i  = w_i rho [1 - 1/2 (u.u)/cs_square + (ci . u / cs_square) + (1/2) (ci . u / cs_square)^2]
	// feq_i  = w_i rho [1 - 1/2 (u.u)/cs_square + (ci . u/cs_square){ 1 + (1/2) (ci . u/cs_square) }]
	// for charge transport equation, just change u into u + KE
	// heq_i  = w_i charge [1 - 1/2 (u.u)/cs_square + (ci . u/cs_square){ 1 + (1/2) (ci . u/cs_square) }]

	// choices of c
	// cx = [0, 1, 0, -1, 0, 1, -1, -1, 1] / CFL
	// cy = [0, 0, 1, 0, -1, 1, 1, -1, -1] / CFL

	// calculate equilibrium
	// temporary variables
	double w0r = w0*rho;
	double wsr = ws*rho;
	double war = wa*rho;
	double wdr = wd*rho;

	double w0c = w0*charge;
	double wsc = ws*charge;
	double wac = wa*charge;
	double wdc = wd*charge;

	double w0cn = w0*chargen;
	double wscn = ws*chargen;
	double wacn = wa*chargen;
	double wdcn = wd*chargen;

	double w0T = w0*temp;
	double wsT = ws*temp;
	double waT = wa*temp;
	double wdT = wd*temp;

	double omusq = 1.0 - 0.5*(ux*ux + uy*uy + uz*uz) / cs_square;
	double omusq_c = 1.0 - 0.5*((ux + K*Ex)*(ux + K*Ex) + (uy + K*Ey)*(uy + K*Ey) + (uz + K*Ez)*(uz + K*Ez)) / cs_square;
	double omusq_cn = 1.0 - 0.5*((ux + Kn*Ex)*(ux + Kn*Ex) + (uy + Kn*Ey)*(uy + Kn*Ey) + (uz + Kn*Ez)*(uz + Kn*Ez)) / cs_square;

	double tux = ux / cs_square / CFL;
	double tuy = uy / cs_square / CFL;
	double tuz = uz / cs_square / CFL;
	double tux_c = (ux + K*Ex) / cs_square / CFL;
	double tuy_c = (uy + K*Ey) / cs_square / CFL;
	double tuz_c = (uz + K*Ez) / cs_square / CFL;
	double tux_cn = (ux + Kn*Ex) / cs_square / CFL;
	double tuy_cn = (uy + Kn*Ey) / cs_square / CFL;
	double tuz_cn = (uz + Kn*Ez) / cs_square / CFL;

	// zero weight
	double fe0 = w0r*(omusq);
	double he0 = w0c*(omusq_c);
	double hne0 = w0cn*(omusq_cn);
	double tempe0 = w0T*(omusq);

	// adjacent weight
	// flow
	double cidot3u = tux;
	double fe1 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux;
	double fe2 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy;
	double fe3 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy;
	double fe4 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz;
	double fe5 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz;
	double fe6 = wsr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// charge
	cidot3u = tux_c;
	double he1 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c;
	double he2 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c;
	double he3 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c;
	double he4 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c;
	double he5 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz_c;
	double he6 = wsc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn;
	double hne1 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn;
	double hne2 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn;
	double hne3 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn;
	double hne4 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn;
	double hne5 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz_cn;
	double hne6 = wscn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	// temperature
	cidot3u = tux;
	double tempe1 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux;
	double tempe2 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy;
	double tempe3 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy;
	double tempe4 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz;
	double tempe5 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuz;
	double tempe6 = wsT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// diagonal weight
	// flow
	cidot3u = tux + tuy;
	double fe7 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tux;
	double fe8 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz;
	double fe9 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux - tuz;
	double fe10 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz + tuy;
	double fe11 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tuz;
	double fe12 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy;
	double fe13 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux;
	double fe14 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuz;
	double fe15 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux;
	double fe16 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tuz;
	double fe17 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tuy;
	double fe18 = war*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// charge
	cidot3u = tux_c + tuy_c;
	double he7 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c - tux_c;
	double he8 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuz_c;
	double he9 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c - tuz_c;
	double he10 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c + tuz_c;
	double he11 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_c - tuz_c;
	double he12 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuy_c;
	double he13 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tux_c;
	double he14 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuz_c;
	double he15 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tux_c;
	double he16 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tuz_c;
	double he17 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tuy_c;
	double he18 = wac*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn + tuy_cn;
	double hne7 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn - tux_cn;
	double hne8 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuz_cn;
	double hne9 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn - tuz_cn;
	double hne10 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn + tuz_cn;
	double hne11 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy_cn - tuz_cn;
	double hne12 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuy_cn;
	double hne13 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tux_cn;
	double hne14 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuz_cn;
	double hne15 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tux_cn;
	double hne16 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tuz_cn;
	double hne17 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tuy_cn;
	double hne18 = wacn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	// temperature
	cidot3u = tux + tuy;
	double tempe7 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tux;
	double tempe8 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz;
	double tempe9 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux - tuz;
	double tempe10 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz;
	double tempe11 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tuz;
	double tempe12 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy;
	double tempe13 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux;
	double tempe14 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuz;
	double tempe15 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux;
	double tempe16 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tuz;
	double tempe17 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tuy;
	double tempe18 = waT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// 3d diagonal
	//flow
	cidot3u = tux + tuy + tuz;
	double fe19 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tuy - tux - tuz;
	double fe20 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuy - tuz;
	double fe21 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux - tuy;
	double fe22 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz - tuy;
	double fe23 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux - tuz;
	double fe24 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz - tux;
	double fe25 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy - tuz;
	double fe26 = wdr*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	//charge
	cidot3u = tux_c + tuy_c + tuz_c;
	double he19 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_c - tuy_c - tuz_c;
	double he20 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuy_c - tuz_c;
	double he21 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_c - tux_c - tuy_c;
	double he22 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c + tuz_c - tuy_c;
	double he23 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c - tux_c - tuz_c;
	double he24 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_c + tuz_c - tux_c;
	double he25 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_c - tuy_c - tuz_c;
	double he26 = wdc*(omusq_c + cidot3u*(1.0 + 0.5*cidot3u));

	// negative charge
	cidot3u = tux_cn + tuy_cn + tuz_cn;
	double hne19 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux_cn - tuy_cn - tuz_cn;
	double hne20 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuy_cn - tuz_cn;
	double hne21 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz_cn - tux_cn - tuy_cn;
	double hne22 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn + tuz_cn - tuy_cn;
	double hne23 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn - tux_cn - tuz_cn;
	double hne24 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy_cn + tuz_cn - tux_cn;
	double hne25 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux_cn - tuy_cn - tuz_cn;
	double hne26 = wdcn*(omusq_cn + cidot3u*(1.0 + 0.5*cidot3u));

	//temperature
	cidot3u = tux + tuy + tuz;
	double tempe19 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = -tux - tuy - tuz;
	double tempe20 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuy - tuz;
	double tempe21 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuz - tux - tuy;
	double tempe22 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux + tuz - tuy;
	double tempe23 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy - tux - tuz;
	double tempe24 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tuy + tuz - tux;
	double tempe25 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));
	cidot3u = tux - tuy - tuz;
	double tempe26 = wdT*(omusq + cidot3u*(1.0 + 0.5*cidot3u));

	// calculate force population
	// temperory variables
	double coe0 = w0 / cs_square;
	double coes = ws / cs_square;
	double coea = wa / cs_square;
	double coed = wd / cs_square;

	double cflinv = 1.0 / CFL;

	double fpop0 = -coe0*(ux*forcex + uy*forcey + uz*forcez);
	double cflinv2 = cflinv*cflinv / cs_square;

	double fpop1 = coes*(-uy*forcey - uz*forcez + ((cflinv - ux) + (cflinv2*ux))*forcex);
	double fpop2 = coes*(-uy*forcey - uz*forcez + ((-cflinv - ux) + (cflinv2*ux))*forcex);
	double fpop3 = coes*(-ux*forcex - uz*forcez + ((cflinv - uy) + (cflinv2*uy))*forcey);
	double fpop4 = coes*(-ux*forcex - uz*forcez + ((-cflinv - uy) + (cflinv2*uy))*forcey);
	double fpop5 = coes*(-ux*forcex - uy*forcey + ((cflinv - uz) + (cflinv2*uz))*forcez);
	double fpop6 = coes*(-ux*forcex - uy*forcey + ((-cflinv - uz) + (cflinv2*uz))*forcez);

	double fpop7  = coea*(((cflinv - ux)  + (ux + uy)*cflinv2)*forcex + ((cflinv - uy)  + (ux + uy)*cflinv2)*forcey - uz*forcez);
	double fpop8  = coea*(((-cflinv - ux) + (ux + uy)*cflinv2)*forcex + ((-cflinv - uy) + (ux + uy)*cflinv2)*forcey - uz*forcez);
	double fpop9  = coea*(((cflinv - ux)  + (ux + uz)*cflinv2)*forcex + ((cflinv - uz)  + (ux + uz)*cflinv2)*forcez - uy*forcey);
	double fpop10 = coea*(((-cflinv - ux) + (ux + uz)*cflinv2)*forcex + ((-cflinv - uz) + (ux + uz)*cflinv2)*forcez - uy*forcey);
	double fpop11 = coea*(((cflinv - uy)  + (uy + uz)*cflinv2)*forcey + ((cflinv - uz)  + (uy + uz)*cflinv2)*forcez - ux*forcex);
	double fpop12 = coea*(((-cflinv - uy) + (uy + uz)*cflinv2)*forcey + ((-cflinv - uz) + (uy + uz)*cflinv2)*forcez - ux*forcex);

	double fpop13 = coea*(((cflinv - ux) + (ux - uy)*cflinv2)*forcex + ((-cflinv - uy) + (-ux + uy)*cflinv2)*forcey - uz*forcez);
	double fpop14 = coea*(((-cflinv - ux) + (ux - uy)*cflinv2)*forcex + ((cflinv - uy) + (-ux + uy)*cflinv2)*forcey - uz*forcez);
	double fpop15 = coea*(((cflinv - ux) + (ux - uz)*cflinv2)*forcex + ((-cflinv - uz) + (-ux + uz)*cflinv2)*forcez - uy*forcey);
	double fpop16 = coea*(((-cflinv - ux) + (ux - uz)*cflinv2)*forcex + ((cflinv - uz) + (-ux + uz)*cflinv2)*forcez - uy*forcey);
	double fpop17 = coea*(((cflinv - uy) + (uy - uz)*cflinv2)*forcey + ((-cflinv - uz) + (-uy + uz)*cflinv2)*forcez - ux*forcex);
	double fpop18 = coea*(((-cflinv - uy) + (uy - uz)*cflinv2)*forcey + ((cflinv - uz) + (-uy + uz)*cflinv2)*forcez - ux*forcex);

	double fpop19 = coed*(((cflinv-ux)+(ux+uy+uz)*cflinv2)*forcex+((cflinv-uy)+(ux+uy+uz)*cflinv2)*forcey+((cflinv-uz)+(ux+uy+uz)*cflinv2)*forcez);
	double fpop20 = coed*(((-cflinv-ux)+(ux+uy+uz)*cflinv2)*forcex+((-cflinv-uy)+(ux+uy+uz)*cflinv2)*forcey+((-cflinv-uz)+(ux+uy+uz)*cflinv2)*forcez);
	double fpop21 = coed*(((cflinv-ux)+(ux+uy-uz)*cflinv2)*forcex+((cflinv-uy)+(ux+uy-uz)*cflinv2)*forcey+((-cflinv-uz)+(-ux-uy+uz)*cflinv2)*forcez);
	double fpop22 = coed*(((-cflinv-ux)+(ux+uy-uz)*cflinv2)*forcex+((-cflinv-uy)+(ux+uy-uz)*cflinv2)*forcey+((cflinv-uz)+(-ux-uy+uz)*cflinv2)*forcez);
	double fpop23 = coed*(((cflinv-ux)+(ux-uy+uz)*cflinv2)*forcex+((-cflinv-uy)+(-ux+uy-uz)*cflinv2)*forcey+((cflinv-uz)+(ux-uy+uz)*cflinv2)*forcez);
	double fpop24 = coed*(((-cflinv-ux)+(ux-uy+uz)*cflinv2)*forcex+((cflinv-uy)+(-ux+uy-uz)*cflinv2)*forcey+((-cflinv-uz)+(ux-uy+uz)*cflinv2)*forcez);
	double fpop25 = coed*(((-cflinv-ux)+(ux-uy-uz)*cflinv2)*forcex+((cflinv-uy)+(-ux+uy+uz)*cflinv2)*forcey+((cflinv-uz)+(-ux+uy+uz)*cflinv2)*forcez);
	double fpop26 = coed*(((cflinv-ux)+(ux-uy-uz)*cflinv2)*forcex+((-cflinv-uy)+(-ux+uy+uz)*cflinv2)*forcey+((-cflinv-uz)+(-ux+uy+uz)*cflinv2)*forcez);
	
	// calculate f1 plus and minus
	double fp0 = ft0;
	double fp1 = 0.5*(ft1 + ft2);
	double fp2 = fp1;
	double fp3 = 0.5*(ft3 + ft4);
	double fp4 = fp3;
	double fp5 = 0.5*(ft5 + ft6);
	double fp6 = fp5;
	double fp7 = 0.5*(ft7 + ft8);
	double fp8 = fp7;
	double fp9 = 0.5*(ft9 + ft10);
	double fp10 = fp9;
	double fp11 = 0.5*(ft11 + ft12);
	double fp12 = fp11;
	double fp13 = 0.5*(ft13 + ft14);
	double fp14 = fp13;
	double fp15 = 0.5*(ft15 + ft16);
	double fp16 = fp15;
	double fp17 = 0.5*(ft17 + ft18);
	double fp18 = fp17;
	double fp19 = 0.5*(ft19 + ft20);
	double fp20 = fp19;
	double fp21 = 0.5*(ft21 + ft22);
	double fp22 = fp21;
	double fp23 = 0.5*(ft23 + ft24);
	double fp24 = fp23;
	double fp25 = 0.5*(ft25 + ft26);
	double fp26 = fp25;

	double fm0 = 0.0;
	double fm1 = 0.5*(ft1 - ft2);
	double fm2 = -fm1;
	double fm3 = 0.5*(ft3 - ft4);
	double fm4 = -fm3;
	double fm5 = 0.5*(ft5 - ft6);
	double fm6 = -fm5;
	double fm7 = 0.5*(ft7 - ft8);
	double fm8 = -fm7;
	double fm9 = 0.5*(ft9 - ft10);
	double fm10 = -fm9;
	double fm11 = 0.5*(ft11 - ft12);
	double fm12 = -fm11;
	double fm13 = 0.5*(ft13 - ft14);
	double fm14 = -fm13;
	double fm15 = 0.5*(ft15 - ft16);
	double fm16 = -fm15;
	double fm17 = 0.5*(ft17 - ft18);
	double fm18 = -fm17;
	double fm19 = 0.5*(ft19 - ft20);
	double fm20 = -fm19;
	double fm21 = 0.5*(ft21 - ft22);
	double fm22 = -fm21;
	double fm23 = 0.5*(ft23 - ft24);
	double fm24 = -fm23;
	double fm25 = 0.5*(ft25 - ft26);
	double fm26 = -fm25;

	// calculate feq plus and minus
	double fep0 = fe0;
	double fep1 = 0.5*(fe1 + fe2);
	double fep2 = fep1;
	double fep3 = 0.5*(fe3 + fe4);
	double fep4 = fep3;
	double fep5 = 0.5*(fe5 + fe6);
	double fep6 = fep5;
	double fep7 = 0.5*(fe7 + fe8);
	double fep8 = fep7;
	double fep9 = 0.5*(fe9 + fe10);
	double fep10 = fep9;
	double fep11 = 0.5*(fe11 + fe12);
	double fep12 = fep11;
	double fep13 = 0.5*(fe13 + fe14);
	double fep14 = fep13;
	double fep15 = 0.5*(fe15 + fe16);
	double fep16 = fep15;
	double fep17 = 0.5*(fe17 + fe18);
	double fep18 = fep17;
	double fep19 = 0.5*(fe19 + fe20);
	double fep20 = fep19;
	double fep21 = 0.5*(fe21 + fe22);
	double fep22 = fep21;
	double fep23 = 0.5*(fe23 + fe24);
	double fep24 = fep23;
	double fep25 = 0.5*(fe25 + fe26);
	double fep26 = fep25;

	double fem0 = 0.0;
	double fem1 = 0.5*(fe1 - fe2);
	double fem2 = -fem1;
	double fem3 = 0.5*(fe3 - fe4);
	double fem4 = -fem3;
	double fem5 = 0.5*(fe5 - fe6);
	double fem6 = -fem5;
	double fem7 = 0.5*(fe7 - fe8);
	double fem8 = -fem7;
	double fem9 = 0.5*(fe9 - fe10);
	double fem10 = -fem9;
	double fem11 = 0.5*(fe11 - fe12);
	double fem12 = -fem11;
	double fem13 = 0.5*(fe13 - fe14);
	double fem14 = -fem13;
	double fem15 = 0.5*(fe15 - fe16);
	double fem16 = -fem15;
	double fem17 = 0.5*(fe17 - fe18);
	double fem18 = -fem17;
	double fem19 = 0.5*(fe19 - fe20);
	double fem20 = -fem19;
	double fem21 = 0.5*(fe21 - fe22);
	double fem22 = -fem21;
	double fem23 = 0.5*(fe23 - fe24);
	double fem24 = -fem23;
	double fem25 = 0.5*(fe25 - fe26);
	double fem26 = -fem25;

	// calculate h1 plus and minus
	double hp0 = ht0;
	double hp1 = 0.5*(ht1 + ht2);
	double hp2 = hp1;
	double hp3 = 0.5*(ht3 + ht4);
	double hp4 = hp3;
	double hp5 = 0.5*(ht5 + ht6);
	double hp6 = hp5;
	double hp7 = 0.5*(ht7 + ht8);
	double hp8 = hp7;
	double hp9 = 0.5*(ht9 + ht10);
	double hp10 = hp9;
	double hp11 = 0.5*(ht11 + ht12);
	double hp12 = hp11;
	double hp13 = 0.5*(ht13 + ht14);
	double hp14 = hp13;
	double hp15 = 0.5*(ht15 + ht16);
	double hp16 = hp15;
	double hp17 = 0.5*(ht17 + ht18);
	double hp18 = hp17;
	double hp19 = 0.5*(ht19 + ht20);
	double hp20 = hp19;
	double hp21 = 0.5*(ht21 + ht22);
	double hp22 = hp21;
	double hp23 = 0.5*(ht23 + ht24);
	double hp24 = hp23;
	double hp25 = 0.5*(ht25 + ht26);
	double hp26 = hp25;

	double hm0 = 0.0;
	double hm1 = 0.5*(ht1 - ht2);
	double hm2 = -hm1;
	double hm3 = 0.5*(ht3 - ht4);
	double hm4 = -hm3;
	double hm5 = 0.5*(ht5 - ht6);
	double hm6 = -hm5;
	double hm7 = 0.5*(ht7 - ht8);
	double hm8 = -hm7;
	double hm9 = 0.5*(ht9 - ht10);
	double hm10 = -hm9;
	double hm11 = 0.5*(ht11 - ht12);
	double hm12 = -hm11;
	double hm13 = 0.5*(ht13 - ht14);
	double hm14 = -hm13;
	double hm15 = 0.5*(ht15 - ht16);
	double hm16 = -hm15;
	double hm17 = 0.5*(ht17 - ht18);
	double hm18 = -hm17;
	double hm19 = 0.5*(ht19 - ht20);
	double hm20 = -hm19;
	double hm21 = 0.5*(ht21 - ht22);
	double hm22 = -hm21;
	double hm23 = 0.5*(ht23 - ht24);
	double hm24 = -hm23;
	double hm25 = 0.5*(ht25 - ht26);
	double hm26 = -hm25;

	// calculate heq plus and minus
	double hep0 = he0;
	double hep1 = 0.5*(he1 + he2);
	double hep2 = hep1;
	double hep3 = 0.5*(he3 + he4);
	double hep4 = hep3;
	double hep5 = 0.5*(he5 + he6);
	double hep6 = hep5;
	double hep7 = 0.5*(he7 + he8);
	double hep8 = hep7;
	double hep9 = 0.5*(he9 + he10);
	double hep10 = hep9;
	double hep11 = 0.5*(he11 + he12);
	double hep12 = hep11;
	double hep13 = 0.5*(he13 + he14);
	double hep14 = hep13;
	double hep15 = 0.5*(he15 + he16);
	double hep16 = hep15;
	double hep17 = 0.5*(he17 + he18);
	double hep18 = hep17;
	double hep19 = 0.5*(he19 + he20);
	double hep20 = hep19;
	double hep21 = 0.5*(he21 + he22);
	double hep22 = hep21;
	double hep23 = 0.5*(he23 + he24);
	double hep24 = hep23;
	double hep25 = 0.5*(he25 + he26);
	double hep26 = hep25;

	double hem0 = 0.0;
	double hem1 = 0.5*(he1 - he2);
	double hem2 = -hem1;
	double hem3 = 0.5*(he3 - he4);
	double hem4 = -hem3;
	double hem5 = 0.5*(he5 - he6);
	double hem6 = -hem5;
	double hem7 = 0.5*(he7 - he8);
	double hem8 = -hem7;
	double hem9 = 0.5*(he9 - he10);
	double hem10 = -hem9;
	double hem11 = 0.5*(he11 - he12);
	double hem12 = -hem11;
	double hem13 = 0.5*(he13 - he14);
	double hem14 = -hem13;
	double hem15 = 0.5*(he15 - he16);
	double hem16 = -hem15;
	double hem17 = 0.5*(he17 - he18);
	double hem18 = -hem17;
	double hem19 = 0.5*(he19 - he20);
	double hem20 = -hem19;
	double hem21 = 0.5*(he21 - he22);
	double hem22 = -hem21;
	double hem23 = 0.5*(he23 - he24);
	double hem24 = -hem23;
	double hem25 = 0.5*(he25 - he26);
	double hem26 = -hem25;

	// calculate hn1 plus and minus
	double hnp0 = hnt0;
	double hnp1 = 0.5*(hnt1 + hnt2);
	double hnp2 = hnp1;
	double hnp3 = 0.5*(hnt3 + hnt4);
	double hnp4 = hnp3;
	double hnp5 = 0.5*(hnt5 + hnt6);
	double hnp6 = hnp5;
	double hnp7 = 0.5*(hnt7 + hnt8);
	double hnp8 = hnp7;
	double hnp9 = 0.5*(hnt9 + hnt10);
	double hnp10 = hnp9;
	double hnp11 = 0.5*(hnt11 + hnt12);
	double hnp12 = hnp11;
	double hnp13 = 0.5*(hnt13 + hnt14);
	double hnp14 = hnp13;
	double hnp15 = 0.5*(hnt15 + hnt16);
	double hnp16 = hnp15;
	double hnp17 = 0.5*(hnt17 + hnt18);
	double hnp18 = hnp17;
	double hnp19 = 0.5*(hnt19 + hnt20);
	double hnp20 = hnp19;
	double hnp21 = 0.5*(hnt21 + hnt22);
	double hnp22 = hnp21;
	double hnp23 = 0.5*(hnt23 + hnt24);
	double hnp24 = hnp23;
	double hnp25 = 0.5*(hnt25 + hnt26);
	double hnp26 = hnp25;

	double hnm0 = 0.0;
	double hnm1 = 0.5*(hnt1 - hnt2);
	double hnm2 = -hnm1;
	double hnm3 = 0.5*(hnt3 - hnt4);
	double hnm4 = -hnm3;
	double hnm5 = 0.5*(hnt5 - hnt6);
	double hnm6 = -hnm5;
	double hnm7 = 0.5*(hnt7 - hnt8);
	double hnm8 = -hnm7;
	double hnm9 = 0.5*(hnt9 - hnt10);
	double hnm10 = -hnm9;
	double hnm11 = 0.5*(hnt11 - hnt12);
	double hnm12 = -hnm11;
	double hnm13 = 0.5*(hnt13 - hnt14);
	double hnm14 = -hnm13;
	double hnm15 = 0.5*(hnt15 - hnt16);
	double hnm16 = -hnm15;
	double hnm17 = 0.5*(hnt17 - hnt18);
	double hnm18 = -hnm17;
	double hnm19 = 0.5*(hnt19 - hnt20);
	double hnm20 = -hnm19;
	double hnm21 = 0.5*(hnt21 - hnt22);
	double hnm22 = -hnm21;
	double hnm23 = 0.5*(hnt23 - hnt24);
	double hnm24 = -hnm23;
	double hnm25 = 0.5*(hnt25 - hnt26);
	double hnm26 = -hnm25;

	// calculate hneq plus and minus
	double hnep0 = hne0;
	double hnep1 = 0.5*(hne1 + hne2);
	double hnep2 = hnep1;
	double hnep3 = 0.5*(hne3 + hne4);
	double hnep4 = hnep3;
	double hnep5 = 0.5*(hne5 + hne6);
	double hnep6 = hnep5;
	double hnep7 = 0.5*(hne7 + hne8);
	double hnep8 = hnep7;
	double hnep9 = 0.5*(hne9 + hne10);
	double hnep10 = hnep9;
	double hnep11 = 0.5*(hne11 + hne12);
	double hnep12 = hnep11;
	double hnep13 = 0.5*(hne13 + hne14);
	double hnep14 = hnep13;
	double hnep15 = 0.5*(hne15 + hne16);
	double hnep16 = hnep15;
	double hnep17 = 0.5*(hne17 + hne18);
	double hnep18 = hnep17;
	double hnep19 = 0.5*(hne19 + hne20);
	double hnep20 = hnep19;
	double hnep21 = 0.5*(hne21 + hne22);
	double hnep22 = hnep21;
	double hnep23 = 0.5*(hne23 + hne24);
	double hnep24 = hnep23;
	double hnep25 = 0.5*(hne25 + hne26);
	double hnep26 = hnep25;

	double hnem0 = 0.0;
	double hnem1 = 0.5*(hne1 - hne2);
	double hnem2 = -hnem1;
	double hnem3 = 0.5*(hne3 - hne4);
	double hnem4 = -hnem3;
	double hnem5 = 0.5*(hne5 - hne6);
	double hnem6 = -hnem5;
	double hnem7 = 0.5*(hne7 - hne8);
	double hnem8 = -hnem7;
	double hnem9 = 0.5*(hne9 - hne10);
	double hnem10 = -hnem9;
	double hnem11 = 0.5*(hne11 - hne12);
	double hnem12 = -hnem11;
	double hnem13 = 0.5*(hne13 - hne14);
	double hnem14 = -hnem13;
	double hnem15 = 0.5*(hne15 - hne16);
	double hnem16 = -hnem15;
	double hnem17 = 0.5*(hne17 - hne18);
	double hnem18 = -hnem17;
	double hnem19 = 0.5*(hne19 - hne20);
	double hnem20 = -hnem19;
	double hnem21 = 0.5*(hne21 - hne22);
	double hnem22 = -hnem21;
	double hnem23 = 0.5*(hne23 - hne24);
	double hnem24 = -hnem23;
	double hnem25 = 0.5*(hne25 - hne26);
	double hnem26 = -hnem25;

	// calculate temp1 plus and minus
	double tempp0 = tempt0;
	double tempp1 = 0.5*(tempt1 + tempt2);
	double tempp2 = tempp1;
	double tempp3 = 0.5*(tempt3 + tempt4);
	double tempp4 = tempp3;
	double tempp5 = 0.5*(tempt5 + tempt6);
	double tempp6 = tempp5;
	double tempp7 = 0.5*(tempt7 + tempt8);
	double tempp8 = tempp7;
	double tempp9 = 0.5*(tempt9 + tempt10);
	double tempp10 = tempp9;
	double tempp11 = 0.5*(tempt11 + tempt12);
	double tempp12 = tempp11;
	double tempp13 = 0.5*(tempt13 + tempt14);
	double tempp14 = tempp13;
	double tempp15 = 0.5*(tempt15 + tempt16);
	double tempp16 = tempp15;
	double tempp17 = 0.5*(tempt17 + tempt18);
	double tempp18 = tempp17;
	double tempp19 = 0.5*(tempt19 + tempt20);
	double tempp20 = tempp19;
	double tempp21 = 0.5*(tempt21 + tempt22);
	double tempp22 = tempp21;
	double tempp23 = 0.5*(tempt23 + tempt24);
	double tempp24 = tempp23;
	double tempp25 = 0.5*(tempt25 + tempt26);
	double tempp26 = tempp25;

	double tempm0 = 0.0;
	double tempm1 = 0.5*(tempt1 - tempt2);
	double tempm2 = -tempm1;
	double tempm3 = 0.5*(tempt3 - tempt4);
	double tempm4 = -tempm3;
	double tempm5 = 0.5*(tempt5 - tempt6);
	double tempm6 = -tempm5;
	double tempm7 = 0.5*(tempt7 - tempt8);
	double tempm8 = -tempm7;
	double tempm9 = 0.5*(tempt9 - tempt10);
	double tempm10 = -tempm9;
	double tempm11 = 0.5*(tempt11 - tempt12);
	double tempm12 = -tempm11;
	double tempm13 = 0.5*(tempt13 - tempt14);
	double tempm14 = -tempm13;
	double tempm15 = 0.5*(tempt15 - tempt16);
	double tempm16 = -tempm15;
	double tempm17 = 0.5*(tempt17 - tempt18);
	double tempm18 = -tempm17;
	double tempm19 = 0.5*(tempt19 - tempt20);
	double tempm20 = -tempm19;
	double tempm21 = 0.5*(tempt21 - tempt22);
	double tempm22 = -tempm21;
	double tempm23 = 0.5*(tempt23 - tempt24);
	double tempm24 = -tempm23;
	double tempm25 = 0.5*(tempt25 - tempt26);
	double tempm26 = -tempm25;

	// calculate tempeq plus and minus
	double tempep0 = tempe0;
	double tempep1 = 0.5*(tempe1 + tempe2);
	double tempep2 = tempep1;
	double tempep3 = 0.5*(tempe3 + tempe4);
	double tempep4 = tempep3;
	double tempep5 = 0.5*(tempe5 + tempe6);
	double tempep6 = tempep5;
	double tempep7 = 0.5*(tempe7 + tempe8);
	double tempep8 = tempep7;
	double tempep9 = 0.5*(tempe9 + tempe10);
	double tempep10 = tempep9;
	double tempep11 = 0.5*(tempe11 + tempe12);
	double tempep12 = tempep11;
	double tempep13 = 0.5*(tempe13 + tempe14);
	double tempep14 = tempep13;
	double tempep15 = 0.5*(tempe15 + tempe16);
	double tempep16 = tempep15;
	double tempep17 = 0.5*(tempe17 + tempe18);
	double tempep18 = tempep17;
	double tempep19 = 0.5*(tempe19 + tempe20);
	double tempep20 = tempep19;
	double tempep21 = 0.5*(tempe21 + tempe22);
	double tempep22 = tempep21;
	double tempep23 = 0.5*(tempe23 + tempe24);
	double tempep24 = tempep23;
	double tempep25 = 0.5*(tempe25 + tempe26);
	double tempep26 = tempep25;

	double tempem0 = 0.0;
	double tempem1 = 0.5*(tempe1 - tempe2);
	double tempem2 = -tempem1;
	double tempem3 = 0.5*(tempe3 - tempe4);
	double tempem4 = -tempem3;
	double tempem5 = 0.5*(tempe5 - tempe6);
	double tempem6 = -tempem5;
	double tempem7 = 0.5*(tempe7 - tempe8);
	double tempem8 = -tempem7;
	double tempem9 = 0.5*(tempe9 - tempe10);
	double tempem10 = -tempem9;
	double tempem11 = 0.5*(tempe11 - tempe12);
	double tempem12 = -tempem11;
	double tempem13 = 0.5*(tempe13 - tempe14);
	double tempem14 = -tempem13;
	double tempem15 = 0.5*(tempe15 - tempe16);
	double tempem16 = -tempem15;
	double tempem17 = 0.5*(tempe17 - tempe18);
	double tempem18 = -tempem17;
	double tempem19 = 0.5*(tempe19 - tempe20);
	double tempem20 = -tempem19;
	double tempem21 = 0.5*(tempe21 - tempe22);
	double tempem22 = -tempem21;
	double tempem23 = 0.5*(tempe23 - tempe24);
	double tempem24 = -tempem23;
	double tempem25 = 0.5*(tempe25 - tempe26);
	double tempem26 = -tempem25;
	
	// calculate force_plus and force_minus
	double forcep0 = fpop0;
	double forcep1 = 0.5*(fpop1 + fpop2);
	double forcep2 = forcep1;
	double forcep3 = 0.5*(fpop3 + fpop4);
	double forcep4 = forcep3;
	double forcep5 = 0.5*(fpop5 + fpop6);
	double forcep6 = forcep5;
	double forcep7 = 0.5*(fpop7 + fpop8);
	double forcep8 = forcep7;
	double forcep9 = 0.5*(fpop9 + fpop10);
	double forcep10 = forcep9;
	double forcep11 = 0.5*(fpop11 + fpop12);
	double forcep12 = forcep11;
	double forcep13 = 0.5*(fpop13 + fpop14);
	double forcep14 = forcep13;
	double forcep15 = 0.5*(fpop15 + fpop16);
	double forcep16 = forcep15;
	double forcep17 = 0.5*(fpop17 + fpop18);
	double forcep18 = forcep17;
	double forcep19 = 0.5*(fpop19 + fpop20);
	double forcep20 = forcep19;
	double forcep21 = 0.5*(fpop21 + fpop22);
	double forcep22 = forcep21;
	double forcep23 = 0.5*(fpop23 + fpop24);
	double forcep24 = forcep23;
	double forcep25 = 0.5*(fpop25 + fpop26);
	double forcep26 = forcep25;
	
	double forcem0 = 0.0;
	double forcem1 = 0.5*(fpop1 - fpop2);
	double forcem2 = -forcem1;
	double forcem3 = 0.5*(fpop3 - fpop4);
	double forcem4 = -forcem3;
	double forcem5 = 0.5*(fpop5 - fpop6);
	double forcem6 = -forcem5;
	double forcem7 = 0.5*(fpop7 - fpop8);
	double forcem8 = -forcem7;
	double forcem9 = 0.5*(fpop9 - fpop10);
	double forcem10 = -forcem9;
	double forcem11 = 0.5*(fpop11 - fpop12);
	double forcem12 = -forcem11;
	double forcem13 = 0.5*(fpop13 - fpop14);
	double forcem14 = -forcem13;
	double forcem15 = 0.5*(fpop15 - fpop16);
	double forcem16 = -forcem15;
	double forcem17 = 0.5*(fpop17 - fpop18);
	double forcem18 = -forcem17;
	double forcem19 = 0.5*(fpop19 - fpop20);
	double forcem20 = -forcem19;
	double forcem21 = 0.5*(fpop21 - fpop22);
	double forcem22 = -forcem21;
	double forcem23 = 0.5*(fpop23 - fpop24);
	double forcem24 = -forcem23;
	double forcem25 = 0.5*(fpop25 - fpop26);
	double forcem26 = -forcem25;

	double sp = 1.0 - 0.5*dt*omega_plus;
	double sm = 1.0 - 0.5*dt*omega_minus;

	double source0 = sp*fpop0;
	double source1 = sp*forcep1 + sm*forcem1;
	double source2 = sp*forcep2 + sm*forcem2;
	double source3 = sp*forcep3 + sm*forcem3;
	double source4 = sp*forcep4 + sm*forcem4;
	double source5 = sp*forcep5 + sm*forcem5;
	double source6 = sp*forcep6 + sm*forcem6;
	double source7 = sp*forcep7 + sm*forcem7;
	double source8 = sp*forcep8 + sm*forcem8;
	double source9 = sp*forcep9 + sm*forcem9;
	double source10 = sp*forcep10 + sm*forcem10;
	double source11 = sp*forcep11 + sm*forcem11;
	double source12 = sp*forcep12 + sm*forcem12;
	double source13 = sp*forcep13 + sm*forcem13;
	double source14 = sp*forcep14 + sm*forcem14;
	double source15 = sp*forcep15 + sm*forcem15;
	double source16 = sp*forcep16 + sm*forcem16;
	double source17 = sp*forcep17 + sm*forcem17;
	double source18 = sp*forcep18 + sm*forcem18;
	double source19 = sp*forcep19 + sm*forcem19;
	double source20 = sp*forcep20 + sm*forcem20;
	double source21 = sp*forcep21 + sm*forcem21;
	double source22 = sp*forcep22 + sm*forcem22;
	double source23 = sp*forcep23 + sm*forcem23;
	double source24 = sp*forcep24 + sm*forcem24;
	double source25 = sp*forcep25 + sm*forcem25;
	double source26 = sp*forcep26 + sm*forcem26;
	// ===============================================================
	//if (x == 5 && y == 1) {
	//	printf("%2.16g\n", charge);

	//printf("%g\n", source1);

	//}
	// ===============================================================
	// ===============================================================
	// temporary variables (relaxation times)
	double tw0rp = omega_plus*dt;  //   omega_plus*dt 
	double tw0rm = omega_minus*dt; //   omega_minus*dt 
	double tw0cp = omega_c_plus*dt;  //   omega_c_plus*dt 
	double tw0cm = omega_c_minus*dt; //   omega_c_minus*dt 
	double tw0cnp = omega_cn_plus*dt;  //   omega_c_plus*dt 
	double tw0cnm = omega_cn_minus*dt; //   omega_c_minus*dt 
	double tw0Tp = omega_T_plus*dt;  //   omega_T_plus*dt 
	double tw0Tm = omega_T_minus*dt; //   omega_T_minus*dt 

	// TRT collision operations

	f0[gpu_field0_index(x, y, z)] = ft0 - (tw0rp * (fp0 - fep0) + tw0rm * (fm0 - fem0)) + dt*source0;
	h0[gpu_field0_index(x, y, z)] = ht0 - (tw0cp * (hp0 - hep0) + tw0cm * (hm0 - hem0));
	hn0[gpu_field0_index(x, y, z)] = hnt0 - (tw0cnp * (hnp0 - hnep0) + tw0cnm * (hnm0 - hnem0));
	temp0[gpu_field0_index(x, y, z)] = tempt0 - (tw0Tp * (tempp0 - tempep0) + tw0Tm * (tempm0 - tempem0));

	f2[gpu_fieldn_index(x, y, z, 1)] = ft1 - (tw0rp * (fp1 - fep1) + tw0rm * (fm1 - fem1)) + dt*source1;
	h2[gpu_fieldn_index(x, y, z, 1)] = ht1 - (tw0cp * (hp1 - hep1) + tw0cm * (hm1 - hem1));
	hn2[gpu_fieldn_index(x, y, z, 1)] = hnt1 - (tw0cnp * (hnp1 - hnep1) + tw0cnm * (hnm1 - hnem1));
	temp2[gpu_fieldn_index(x, y, z, 1)] = tempt1 - (tw0Tp * (tempp1 - tempep1) + tw0Tm * (tempm1 - tempem1));

	f2[gpu_fieldn_index(x, y, z, 2)] = ft2 - (tw0rp * (fp2 - fep2) + tw0rm * (fm2 - fem2)) + dt*source2;
	h2[gpu_fieldn_index(x, y, z, 2)] = ht2 - (tw0cp * (hp2 - hep2) + tw0cm * (hm2 - hem2));
	hn2[gpu_fieldn_index(x, y, z, 2)] = hnt2 - (tw0cnp * (hnp2 - hnep2) + tw0cnm * (hnm2 - hnem2));
	temp2[gpu_fieldn_index(x, y, z, 2)] = tempt2 - (tw0Tp * (tempp2 - tempep2) + tw0Tm * (tempm2 - tempem2));

	f2[gpu_fieldn_index(x, y, z, 3)] = ft3 - (tw0rp * (fp3 - fep3) + tw0rm * (fm3 - fem3)) + dt*source3;
	h2[gpu_fieldn_index(x, y, z, 3)] = ht3 - (tw0cp * (hp3 - hep3) + tw0cm * (hm3 - hem3));
	hn2[gpu_fieldn_index(x, y, z, 3)] = hnt3 - (tw0cnp * (hnp3 - hnep3) + tw0cnm * (hnm3 - hnem3));
	temp2[gpu_fieldn_index(x, y, z, 3)] = tempt3 - (tw0Tp * (tempp3 - tempep3) + tw0Tm * (tempm3 - tempem3));

	f2[gpu_fieldn_index(x, y, z, 4)] = ft4 - (tw0rp * (fp4 - fep4) + tw0rm * (fm4 - fem4)) + dt*source4;
	h2[gpu_fieldn_index(x, y, z, 4)] = ht4 - (tw0cp * (hp4 - hep4) + tw0cm * (hm4 - hem4));
	hn2[gpu_fieldn_index(x, y, z, 4)] = hnt4 - (tw0cnp * (hnp4 - hnep4) + tw0cnm * (hnm4 - hnem4));
	temp2[gpu_fieldn_index(x, y, z, 4)] = tempt4 - (tw0Tp * (tempp4 - tempep4) + tw0Tm * (tempm4 - tempem4));

	f2[gpu_fieldn_index(x, y, z, 5)] = ft5 - (tw0rp * (fp5 - fep5) + tw0rm * (fm5 - fem5)) + dt*source5;
	h2[gpu_fieldn_index(x, y, z, 5)] = ht5 - (tw0cp * (hp5 - hep5) + tw0cm * (hm5 - hem5));
	hn2[gpu_fieldn_index(x, y, z, 5)] = hnt5 - (tw0cnp * (hnp5 - hnep5) + tw0cnm * (hnm5 - hnem5));
	temp2[gpu_fieldn_index(x, y, z, 5)] = tempt5 - (tw0Tp * (tempp5 - tempep5) + tw0Tm * (tempm5 - tempem5));

	f2[gpu_fieldn_index(x, y, z, 6)] = ft6 - (tw0rp * (fp6 - fep6) + tw0rm * (fm6 - fem6)) + dt*source6;
	h2[gpu_fieldn_index(x, y, z, 6)] = ht6 - (tw0cp * (hp6 - hep6) + tw0cm * (hm6 - hem6));
	hn2[gpu_fieldn_index(x, y, z, 6)] = hnt6 - (tw0cnp * (hnp6 - hnep6) + tw0cnm * (hnm6 - hnem6));
	temp2[gpu_fieldn_index(x, y, z, 6)] = tempt6 - (tw0Tp * (tempp6 - tempep6) + tw0Tm * (tempm6 - tempem6));

	f2[gpu_fieldn_index(x, y, z, 7)] = ft7 - (tw0rp * (fp7 - fep7) + tw0rm * (fm7 - fem7)) + dt*source7;
	h2[gpu_fieldn_index(x, y, z, 7)] = ht7 - (tw0cp * (hp7 - hep7) + tw0cm * (hm7 - hem7));
	hn2[gpu_fieldn_index(x, y, z, 7)] = hnt7 - (tw0cnp * (hnp7 - hnep7) + tw0cnm * (hnm7 - hnem7));
	temp2[gpu_fieldn_index(x, y, z, 7)] = tempt7 - (tw0Tp * (tempp7 - tempep7) + tw0Tm * (tempm7 - tempem7));

	f2[gpu_fieldn_index(x, y, z, 8)] = ft8 - (tw0rp * (fp8 - fep8) + tw0rm * (fm8 - fem8)) + dt*source8;
	h2[gpu_fieldn_index(x, y, z, 8)] = ht8 - (tw0cp * (hp8 - hep8) + tw0cm * (hm8 - hem8));
	hn2[gpu_fieldn_index(x, y, z, 8)] = hnt8 - (tw0cnp * (hnp8 - hnep8) + tw0cnm * (hnm8 - hnem8));
	temp2[gpu_fieldn_index(x, y, z, 8)] = tempt8 - (tw0Tp * (tempp8 - tempep8) + tw0Tm * (tempm8 - tempem8));

	f2[gpu_fieldn_index(x, y, z, 9)] = ft9 - (tw0rp * (fp9 - fep9) + tw0rm * (fm9 - fem9)) + dt*source9;
	h2[gpu_fieldn_index(x, y, z, 9)] = ht9 - (tw0cp * (hp9 - hep9) + tw0cm * (hm9 - hem9));
	hn2[gpu_fieldn_index(x, y, z, 9)] = hnt9 - (tw0cnp * (hnp9 - hnep9) + tw0cnm * (hnm9 - hnem9));
	temp2[gpu_fieldn_index(x, y, z, 9)] = tempt9 - (tw0Tp * (tempp9 - tempep9) + tw0Tm * (tempm9 - tempem9));


	f2[gpu_fieldn_index(x, y, z, 10)] = ft10 - (tw0rp * (fp10 - fep10) + tw0rm * (fm10 - fem10)) + dt*source10;
	h2[gpu_fieldn_index(x, y, z, 10)] = ht10 - (tw0cp * (hp10 - hep10) + tw0cm * (hm10 - hem10));
	hn2[gpu_fieldn_index(x, y, z, 10)] = hnt10 - (tw0cnp * (hnp10 - hnep10) + tw0cnm * (hnm10 - hnem10));
	temp2[gpu_fieldn_index(x, y, z, 10)] = tempt10 - (tw0Tp * (tempp10 - tempep10) + tw0Tm * (tempm10 - tempem10));

	f2[gpu_fieldn_index(x, y, z, 11)] = ft11 - (tw0rp * (fp11 - fep11) + tw0rm * (fm11 - fem11)) + dt*source11;
	h2[gpu_fieldn_index(x, y, z, 11)] = ht11 - (tw0cp * (hp11 - hep11) + tw0cm * (hm11 - hem11));
	hn2[gpu_fieldn_index(x, y, z, 11)] = hnt11 - (tw0cnp * (hnp11 - hnep11) + tw0cnm * (hnm11 - hnem11));
	temp2[gpu_fieldn_index(x, y, z, 11)] = tempt11 - (tw0Tp * (tempp11 - tempep11) + tw0Tm * (tempm11 - tempem11));

	f2[gpu_fieldn_index(x, y, z, 12)] = ft12 - (tw0rp * (fp12 - fep12) + tw0rm * (fm12 - fem12)) + dt*source12;
	h2[gpu_fieldn_index(x, y, z, 12)] = ht12 - (tw0cp * (hp12 - hep12) + tw0cm * (hm12 - hem12));
	hn2[gpu_fieldn_index(x, y, z, 12)] = hnt12 - (tw0cnp * (hnp12 - hnep12) + tw0cnm * (hnm12 - hnem12));
	temp2[gpu_fieldn_index(x, y, z, 12)] = tempt12 - (tw0Tp * (tempp12 - tempep12) + tw0Tm * (tempm12 - tempem12));

	f2[gpu_fieldn_index(x, y, z, 13)] = ft13 - (tw0rp * (fp13 - fep13) + tw0rm * (fm13 - fem13)) + dt*source13;
	h2[gpu_fieldn_index(x, y, z, 13)] = ht13 - (tw0cp * (hp13 - hep13) + tw0cm * (hm13 - hem13));
	hn2[gpu_fieldn_index(x, y, z, 13)] = hnt13 - (tw0cnp * (hnp13 - hnep13) + tw0cnm * (hnm13 - hnem13));
	temp2[gpu_fieldn_index(x, y, z, 13)] = tempt13 - (tw0Tp * (tempp13 - tempep13) + tw0Tm * (tempm13 - tempem13));

	f2[gpu_fieldn_index(x, y, z, 14)] = ft14 - (tw0rp * (fp14 - fep14) + tw0rm * (fm14 - fem14)) + dt*source14;
	h2[gpu_fieldn_index(x, y, z, 14)] = ht14 - (tw0cp * (hp14 - hep14) + tw0cm * (hm14 - hem14));
	hn2[gpu_fieldn_index(x, y, z, 14)] = hnt14 - (tw0cnp * (hnp14 - hnep14) + tw0cnm * (hnm14 - hnem14));
	temp2[gpu_fieldn_index(x, y, z, 14)] = tempt14 - (tw0Tp * (tempp14 - tempep14) + tw0Tm * (tempm14 - tempem14));

	f2[gpu_fieldn_index(x, y, z, 15)] = ft15 - (tw0rp * (fp15 - fep15) + tw0rm * (fm15 - fem15)) + dt*source15;
	h2[gpu_fieldn_index(x, y, z, 15)] = ht15 - (tw0cp * (hp15 - hep15) + tw0cm * (hm15 - hem15));
	hn2[gpu_fieldn_index(x, y, z, 15)] = hnt15 - (tw0cnp * (hnp15 - hnep15) + tw0cnm * (hnm15 - hnem15));
	temp2[gpu_fieldn_index(x, y, z, 15)] = tempt15 - (tw0Tp * (tempp15 - tempep15) + tw0Tm * (tempm15 - tempem15));

	f2[gpu_fieldn_index(x, y, z, 16)] = ft16 - (tw0rp * (fp16 - fep16) + tw0rm * (fm16 - fem16)) + dt*source16;
	h2[gpu_fieldn_index(x, y, z, 16)] = ht16 - (tw0cp * (hp16 - hep16) + tw0cm * (hm16 - hem16));
	hn2[gpu_fieldn_index(x, y, z, 16)] = hnt16 - (tw0cnp * (hnp16 - hnep16) + tw0cnm * (hnm16 - hnem16));
	temp2[gpu_fieldn_index(x, y, z, 16)] = tempt16 - (tw0Tp * (tempp16 - tempep16) + tw0Tm * (tempm16 - tempem16));

	f2[gpu_fieldn_index(x, y, z, 17)] = ft17 - (tw0rp * (fp17 - fep17) + tw0rm * (fm17 - fem17)) + dt*source17;
	h2[gpu_fieldn_index(x, y, z, 17)] = ht17 - (tw0cp * (hp17 - hep17) + tw0cm * (hm17 - hem17));
	hn2[gpu_fieldn_index(x, y, z, 17)] = hnt17 - (tw0cnp * (hnp17 - hnep17) + tw0cnm * (hnm17 - hnem17));
	temp2[gpu_fieldn_index(x, y, z, 17)] = tempt17 - (tw0Tp * (tempp17 - tempep17) + tw0Tm * (tempm17 - tempem17));

	f2[gpu_fieldn_index(x, y, z, 18)] = ft18 - (tw0rp * (fp18 - fep18) + tw0rm * (fm18 - fem18)) + dt*source18;
	h2[gpu_fieldn_index(x, y, z, 18)] = ht18 - (tw0cp * (hp18 - hep18) + tw0cm * (hm18 - hem18));
	hn2[gpu_fieldn_index(x, y, z, 18)] = hnt18 - (tw0cnp * (hnp18 - hnep18) + tw0cnm * (hnm18 - hnem18));
	temp2[gpu_fieldn_index(x, y, z, 18)] = tempt18 - (tw0Tp * (tempp18 - tempep18) + tw0Tm * (tempm18 - tempem18));

	f2[gpu_fieldn_index(x, y, z, 19)] = ft19 - (tw0rp * (fp19 - fep19) + tw0rm * (fm19 - fem19)) + dt*source19;
	h2[gpu_fieldn_index(x, y, z, 19)] = ht19 - (tw0cp * (hp19 - hep19) + tw0cm * (hm19 - hem19));
	hn2[gpu_fieldn_index(x, y, z, 19)] = hnt19 - (tw0cnp * (hnp19 - hnep19) + tw0cnm * (hnm19 - hnem19));
	temp2[gpu_fieldn_index(x, y, z, 19)] = tempt19 - (tw0Tp * (tempp19 - tempep19) + tw0Tm * (tempm19 - tempem19));

	f2[gpu_fieldn_index(x, y, z, 20)] = ft20 - (tw0rp * (fp20 - fep20) + tw0rm * (fm20 - fem20)) + dt*source20;
	h2[gpu_fieldn_index(x, y, z, 20)] = ht20 - (tw0cp * (hp20 - hep20) + tw0cm * (hm20 - hem20));
	hn2[gpu_fieldn_index(x, y, z, 20)] = hnt20 - (tw0cnp * (hnp20 - hnep20) + tw0cnm * (hnm20 - hnem20));
	temp2[gpu_fieldn_index(x, y, z, 20)] = tempt20 - (tw0Tp * (tempp20 - tempep20) + tw0Tm * (tempm20 - tempem20));
	
	f2[gpu_fieldn_index(x, y, z, 21)] = ft21 - (tw0rp * (fp21 - fep21) + tw0rm * (fm21 - fem21)) + dt*source21;
	h2[gpu_fieldn_index(x, y, z, 21)] = ht21 - (tw0cp * (hp21 - hep21) + tw0cm * (hm21 - hem21));
	hn2[gpu_fieldn_index(x, y, z, 21)] = hnt21 - (tw0cnp * (hnp21 - hnep21) + tw0cnm * (hnm21 - hnem21));
	temp2[gpu_fieldn_index(x, y, z, 21)] = tempt21 - (tw0Tp * (tempp21 - tempep21) + tw0Tm * (tempm21 - tempem21));

	f2[gpu_fieldn_index(x, y, z, 22)] = ft22 - (tw0rp * (fp22 - fep22) + tw0rm * (fm22 - fem22)) + dt*source22;
	h2[gpu_fieldn_index(x, y, z, 22)] = ht22 - (tw0cp * (hp22 - hep22) + tw0cm * (hm22 - hem22));
	hn2[gpu_fieldn_index(x, y, z, 22)] = hnt22 - (tw0cnp * (hnp22 - hnep22) + tw0cnm * (hnm22 - hnem22));
	temp2[gpu_fieldn_index(x, y, z, 22)] = tempt22 - (tw0Tp * (tempp22 - tempep22) + tw0Tm * (tempm22 - tempem22));

	f2[gpu_fieldn_index(x, y, z, 23)] = ft23 - (tw0rp * (fp23 - fep23) + tw0rm * (fm23 - fem23)) + dt*source23;
	h2[gpu_fieldn_index(x, y, z, 23)] = ht23 - (tw0cp * (hp23 - hep23) + tw0cm * (hm23 - hem23));
	hn2[gpu_fieldn_index(x, y, z, 23)] = hnt23 - (tw0cnp * (hnp23 - hnep23) + tw0cnm * (hnm23 - hnem23));
	temp2[gpu_fieldn_index(x, y, z, 23)] = tempt23 - (tw0Tp * (tempp23 - tempep23) + tw0Tm * (tempm23 - tempem23));

	f2[gpu_fieldn_index(x, y, z, 24)] = ft24 - (tw0rp * (fp24 - fep24) + tw0rm * (fm24 - fem24)) + dt*source24;
	h2[gpu_fieldn_index(x, y, z, 24)] = ht24 - (tw0cp * (hp24 - hep24) + tw0cm * (hm24 - hem24));
	hn2[gpu_fieldn_index(x, y, z, 24)] = hnt24 - (tw0cnp * (hnp24 - hnep24) + tw0cnm * (hnm24 - hnem24));
	temp2[gpu_fieldn_index(x, y, z, 24)] = tempt24 - (tw0Tp * (tempp24 - tempep24) + tw0Tm * (tempm24 - tempem24));

	f2[gpu_fieldn_index(x, y, z, 25)] = ft25 - (tw0rp * (fp25 - fep25) + tw0rm * (fm25 - fem25)) + dt*source25;
	h2[gpu_fieldn_index(x, y, z, 25)] = ht25 - (tw0cp * (hp25 - hep25) + tw0cm * (hm25 - hem25));
	hn2[gpu_fieldn_index(x, y, z, 25)] = hnt25 - (tw0cnp * (hnp25 - hnep25) + tw0cnm * (hnm25 - hnem25));
	temp2[gpu_fieldn_index(x, y, z, 25)] = tempt25 - (tw0Tp * (tempp25 - tempep25) + tw0Tm * (tempm25 - tempem25));

	f2[gpu_fieldn_index(x, y, z, 26)] = ft26 - (tw0rp * (fp26 - fep26) + tw0rm * (fm26 - fem26)) + dt*source26;
	h2[gpu_fieldn_index(x, y, z, 26)] = ht26 - (tw0cp * (hp26 - hep26) + tw0cm * (hm26 - hem26));
	hn2[gpu_fieldn_index(x, y, z, 26)] = hnt26 - (tw0cnp * (hnp26 - hnep26) + tw0cnm * (hnm26 - hnem26));
	temp2[gpu_fieldn_index(x, y, z, 26)] = tempt26 - (tw0Tp * (tempp26 - tempep26) + tw0Tm * (tempm26 - tempem26));
}

__global__ void gpu_boundary(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *hn0, double *hn1, double *hn2,
	double *temp0, double *temp1, double *temp2, double *f0bc)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;

	// set perturb = 0
	perturb = 0;

	// Full way bounce back
	if (z == 0) {
		// lower plate
		f0[gpu_field0_index(x, y, 0)]    = f0bc[gpu_field0_index(x, y, 0)];
		f2[gpu_fieldn_index(x, y, 0, 1)] = f1[gpu_fieldn_index(x, y, 0, 2)];
		f2[gpu_fieldn_index(x, y, 0, 2)] = f1[gpu_fieldn_index(x, y, 0, 1)];
		f2[gpu_fieldn_index(x, y, 0, 3)] = f1[gpu_fieldn_index(x, y, 0, 4)];
		f2[gpu_fieldn_index(x, y, 0, 4)] = f1[gpu_fieldn_index(x, y, 0, 3)];
		f2[gpu_fieldn_index(x, y, 0, 5)] = f1[gpu_fieldn_index(x, y, 0, 6)];
		f2[gpu_fieldn_index(x, y, 0, 6)] = f1[gpu_fieldn_index(x, y, 0, 5)];
		f2[gpu_fieldn_index(x, y, 0, 7)] = f1[gpu_fieldn_index(x, y, 0, 8)];
		f2[gpu_fieldn_index(x, y, 0, 8)] = f1[gpu_fieldn_index(x, y, 0, 7)];
		f2[gpu_fieldn_index(x, y, 0, 9)] = f1[gpu_fieldn_index(x, y, 0, 10)];
		f2[gpu_fieldn_index(x, y, 0, 10)] = f1[gpu_fieldn_index(x, y, 0, 9)];
		f2[gpu_fieldn_index(x, y, 0, 11)] = f1[gpu_fieldn_index(x, y, 0, 12)];
		f2[gpu_fieldn_index(x, y, 0, 12)] = f1[gpu_fieldn_index(x, y, 0, 11)];
		f2[gpu_fieldn_index(x, y, 0, 13)] = f1[gpu_fieldn_index(x, y, 0, 14)];
		f2[gpu_fieldn_index(x, y, 0, 14)] = f1[gpu_fieldn_index(x, y, 0, 13)];
		f2[gpu_fieldn_index(x, y, 0, 15)] = f1[gpu_fieldn_index(x, y, 0, 16)];
		f2[gpu_fieldn_index(x, y, 0, 16)] = f1[gpu_fieldn_index(x, y, 0, 15)];
		f2[gpu_fieldn_index(x, y, 0, 17)] = f1[gpu_fieldn_index(x, y, 0, 18)];
		f2[gpu_fieldn_index(x, y, 0, 18)] = f1[gpu_fieldn_index(x, y, 0, 17)];
		f2[gpu_fieldn_index(x, y, 0, 19)] = f1[gpu_fieldn_index(x, y, 0, 20)];
		f2[gpu_fieldn_index(x, y, 0, 20)] = f1[gpu_fieldn_index(x, y, 0, 19)];
		f2[gpu_fieldn_index(x, y, 0, 21)] = f1[gpu_fieldn_index(x, y, 0, 22)];
		f2[gpu_fieldn_index(x, y, 0, 22)] = f1[gpu_fieldn_index(x, y, 0, 21)];
		f2[gpu_fieldn_index(x, y, 0, 23)] = f1[gpu_fieldn_index(x, y, 0, 24)];
		f2[gpu_fieldn_index(x, y, 0, 24)] = f1[gpu_fieldn_index(x, y, 0, 23)];
		f2[gpu_fieldn_index(x, y, 0, 25)] = f1[gpu_fieldn_index(x, y, 0, 26)];
		f2[gpu_fieldn_index(x, y, 0, 26)] = f1[gpu_fieldn_index(x, y, 0, 25)];
		return;
	}

	// direction numbering scheme
	// 6 2 5
	// 3 0 1
	// 7 4 8
	// Boundary conditions
	double multis = 2.0*rho0*uw / cs_square * ws / CFL;
	double multia = 2.0*rho0*uw / cs_square * wa / CFL;
	double multid = 2.0*rho0*uw / cs_square * wd / CFL;
	if (z ==  NZ - 1) {
		// upper plate
		f0[gpu_field0_index(x, y, NZ - 1)]    = f0bc[gpu_field0_index(x, y, 1)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 1)] = f1[gpu_fieldn_index(x, y, NZ - 1, 2)] + multis;
		f2[gpu_fieldn_index(x, y, NZ - 1, 2)] = f1[gpu_fieldn_index(x, y, NZ - 1, 1)] - multis;
		f2[gpu_fieldn_index(x, y, NZ - 1, 3)] = f1[gpu_fieldn_index(x, y, NZ - 1, 4)] + multis;
		f2[gpu_fieldn_index(x, y, NZ - 1, 4)] = f1[gpu_fieldn_index(x, y, NZ - 1, 3)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 5)] = f1[gpu_fieldn_index(x, y, NZ - 1, 6)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 6)] = f1[gpu_fieldn_index(x, y, NZ - 1, 5)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 7)] = f1[gpu_fieldn_index(x, y, NZ - 1, 8)] + multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 8)] = f1[gpu_fieldn_index(x, y, NZ - 1, 7)] - multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 9)] = f1[gpu_fieldn_index(x, y, NZ - 1, 10)] + multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 10)] = f1[gpu_fieldn_index(x, y, NZ - 1, 9)] - multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 11)] = f1[gpu_fieldn_index(x, y, NZ - 1, 12)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 12)] = f1[gpu_fieldn_index(x, y, NZ - 1, 11)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 13)] = f1[gpu_fieldn_index(x, y, NZ - 1, 14)] + multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 14)] = f1[gpu_fieldn_index(x, y, NZ - 1, 13)] - multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 15)] = f1[gpu_fieldn_index(x, y, NZ - 1, 16)] + multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 16)] = f1[gpu_fieldn_index(x, y, NZ - 1, 15)] - multia;
		f2[gpu_fieldn_index(x, y, NZ - 1, 17)] = f1[gpu_fieldn_index(x, y, NZ - 1, 18)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 18)] = f1[gpu_fieldn_index(x, y, NZ - 1, 17)];
		f2[gpu_fieldn_index(x, y, NZ - 1, 19)] = f1[gpu_fieldn_index(x, y, NZ - 1, 20)] + multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 20)] = f1[gpu_fieldn_index(x, y, NZ - 1, 19)] - multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 21)] = f1[gpu_fieldn_index(x, y, NZ - 1, 22)] + multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 22)] = f1[gpu_fieldn_index(x, y, NZ - 1, 21)] - multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 23)] = f1[gpu_fieldn_index(x, y, NZ - 1, 24)] + multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 24)] = f1[gpu_fieldn_index(x, y, NZ - 1, 23)] - multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 25)] = f1[gpu_fieldn_index(x, y, NZ - 1, 26)] - multid;
		f2[gpu_fieldn_index(x, y, NZ - 1, 26)] = f1[gpu_fieldn_index(x, y, NZ - 1, 25)] + multid;

		// Zero charge gradient on Ny
		/*
		h0[gpu_field0_index(x, y, NZ - 1)]    = h0[gpu_field0_index(x, y, NZ - 2)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 1)] = h2[gpu_fieldn_index(x, y, NZ - 2, 1)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 2)] = h2[gpu_fieldn_index(x, y, NZ - 2, 2)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 3)] = h2[gpu_fieldn_index(x, y, NZ - 2, 3)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 4)] = h2[gpu_fieldn_index(x, y, NZ - 2, 4)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 5)] = h2[gpu_fieldn_index(x, y, NZ - 2, 5)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 6)] = h2[gpu_fieldn_index(x, y, NZ - 2, 6)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 7)] = h2[gpu_fieldn_index(x, y, NZ - 2, 7)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 8)] = h2[gpu_fieldn_index(x, y, NZ - 2, 8)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 9)] = h2[gpu_fieldn_index(x, y, NZ - 2, 9)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 10)] = h2[gpu_fieldn_index(x, y, NZ - 2, 10)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 11)] = h2[gpu_fieldn_index(x, y, NZ - 2, 11)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 12)] = h2[gpu_fieldn_index(x, y, NZ - 2, 12)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 13)] = h2[gpu_fieldn_index(x, y, NZ - 2, 13)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 14)] = h2[gpu_fieldn_index(x, y, NZ - 2, 14)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 15)] = h2[gpu_fieldn_index(x, y, NZ - 2, 15)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 16)] = h2[gpu_fieldn_index(x, y, NZ - 2, 16)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 17)] = h2[gpu_fieldn_index(x, y, NZ - 2, 17)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 18)] = h2[gpu_fieldn_index(x, y, NZ - 2, 18)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 19)] = h2[gpu_fieldn_index(x, y, NZ - 2, 19)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 20)] = h2[gpu_fieldn_index(x, y, NZ - 2, 20)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 21)] = h2[gpu_fieldn_index(x, y, NZ - 2, 21)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 22)] = h2[gpu_fieldn_index(x, y, NZ - 2, 22)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 23)] = h2[gpu_fieldn_index(x, y, NZ - 2, 23)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 24)] = h2[gpu_fieldn_index(x, y, NZ - 2, 24)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 25)] = h2[gpu_fieldn_index(x, y, NZ - 2, 25)];
		h2[gpu_fieldn_index(x, y, NZ - 1, 26)] = h2[gpu_fieldn_index(x, y, NZ - 2, 26)];
		*/
		return;
	}
}

__global__ void gpu_stream(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *hn0, double *hn1, double *hn2, double *temp0, double *temp1, double *temp2)
{
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	// streaming step
	unsigned int xp1 = (x + 1) % NX;
	unsigned int yp1 = (y + 1) % NY;
	unsigned int zp1 = (z + 1) % NZ;
	unsigned int xm1 = (NX + x - 1) % NX;
	unsigned int ym1 = (NY + y - 1) % NY;
	unsigned int zm1 = (NZ + z - 1) % NZ;
	// direction numbering scheme
	// 6 2 5
	// 3 0 1
	// 7 4 8

	// load populations from adjacent nodes (ft is post-streaming population of f1)
	// flows
	f1[gpu_fieldn_index(x, y, z, 1)] = f2[gpu_fieldn_index(xm1, y, z, 1)];
	f1[gpu_fieldn_index(x, y, z, 2)] = f2[gpu_fieldn_index(xp1, y, z, 2)];
	f1[gpu_fieldn_index(x, y, z, 3)] = f2[gpu_fieldn_index(x, ym1, z, 3)];
	f1[gpu_fieldn_index(x, y, z, 4)] = f2[gpu_fieldn_index(x, yp1, z, 4)];
	f1[gpu_fieldn_index(x, y, z, 5)] = f2[gpu_fieldn_index(x, y, zm1, 5)];
	f1[gpu_fieldn_index(x, y, z, 6)] = f2[gpu_fieldn_index(x, y, zp1, 6)];
	f1[gpu_fieldn_index(x, y, z, 7)] = f2[gpu_fieldn_index(xm1, ym1, z, 7)];
	f1[gpu_fieldn_index(x, y, z, 8)] = f2[gpu_fieldn_index(xp1, yp1, z, 8)];
	f1[gpu_fieldn_index(x, y, z, 9)] = f2[gpu_fieldn_index(xm1, y, zm1, 9)];
	f1[gpu_fieldn_index(x, y, z, 10)] = f2[gpu_fieldn_index(xp1, y, zp1, 10)];
	f1[gpu_fieldn_index(x, y, z, 11)] = f2[gpu_fieldn_index(x, ym1, zm1, 11)];
	f1[gpu_fieldn_index(x, y, z, 12)] = f2[gpu_fieldn_index(x, yp1, zp1, 12)];
	f1[gpu_fieldn_index(x, y, z, 13)] = f2[gpu_fieldn_index(xm1, yp1, z, 13)];
	f1[gpu_fieldn_index(x, y, z, 14)] = f2[gpu_fieldn_index(xp1, ym1, z, 14)];
	f1[gpu_fieldn_index(x, y, z, 15)] = f2[gpu_fieldn_index(xm1, y, zp1, 15)];
	f1[gpu_fieldn_index(x, y, z, 16)] = f2[gpu_fieldn_index(xp1, y, zm1, 16)];
	f1[gpu_fieldn_index(x, y, z, 17)] = f2[gpu_fieldn_index(x, ym1, zp1, 17)];
	f1[gpu_fieldn_index(x, y, z, 18)] = f2[gpu_fieldn_index(x, yp1, zm1, 18)];
	f1[gpu_fieldn_index(x, y, z, 19)] = f2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
	f1[gpu_fieldn_index(x, y, z, 20)] = f2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
	f1[gpu_fieldn_index(x, y, z, 21)] = f2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
	f1[gpu_fieldn_index(x, y, z, 22)] = f2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
	f1[gpu_fieldn_index(x, y, z, 23)] = f2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
	f1[gpu_fieldn_index(x, y, z, 24)] = f2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
	f1[gpu_fieldn_index(x, y, z, 25)] = f2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
	f1[gpu_fieldn_index(x, y, z, 26)] = f2[gpu_fieldn_index(xm1, yp1, zp1, 26)];

	// charges
	h1[gpu_fieldn_index(x, y, z, 1)] = h2[gpu_fieldn_index(xm1, y, z, 1)];
	h1[gpu_fieldn_index(x, y, z, 2)] = h2[gpu_fieldn_index(xp1, y, z, 2)];
	h1[gpu_fieldn_index(x, y, z, 3)] = h2[gpu_fieldn_index(x, ym1, z, 3)];
	h1[gpu_fieldn_index(x, y, z, 4)] = h2[gpu_fieldn_index(x, yp1, z, 4)];
	h1[gpu_fieldn_index(x, y, z, 5)] = h2[gpu_fieldn_index(x, y, zm1, 5)];
	h1[gpu_fieldn_index(x, y, z, 6)] = h2[gpu_fieldn_index(x, y, zp1, 6)];
	h1[gpu_fieldn_index(x, y, z, 7)] = h2[gpu_fieldn_index(xm1, ym1, z, 7)];
	h1[gpu_fieldn_index(x, y, z, 8)] = h2[gpu_fieldn_index(xp1, yp1, z, 8)];
	h1[gpu_fieldn_index(x, y, z, 9)] = h2[gpu_fieldn_index(xm1, y, zm1, 9)];
	h1[gpu_fieldn_index(x, y, z, 10)] = h2[gpu_fieldn_index(xp1, y, zp1, 10)];
	h1[gpu_fieldn_index(x, y, z, 11)] = h2[gpu_fieldn_index(x, ym1, zm1, 11)];
	h1[gpu_fieldn_index(x, y, z, 12)] = h2[gpu_fieldn_index(x, yp1, zp1, 12)];
	h1[gpu_fieldn_index(x, y, z, 13)] = h2[gpu_fieldn_index(xm1, yp1, z, 13)];
	h1[gpu_fieldn_index(x, y, z, 14)] = h2[gpu_fieldn_index(xp1, ym1, z, 14)];
	h1[gpu_fieldn_index(x, y, z, 15)] = h2[gpu_fieldn_index(xm1, y, zp1, 15)];
	h1[gpu_fieldn_index(x, y, z, 16)] = h2[gpu_fieldn_index(xp1, y, zm1, 16)];
	h1[gpu_fieldn_index(x, y, z, 17)] = h2[gpu_fieldn_index(x, ym1, zp1, 17)];
	h1[gpu_fieldn_index(x, y, z, 18)] = h2[gpu_fieldn_index(x, yp1, zm1, 18)];
	h1[gpu_fieldn_index(x, y, z, 19)] = h2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
	h1[gpu_fieldn_index(x, y, z, 20)] = h2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
	h1[gpu_fieldn_index(x, y, z, 21)] = h2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
	h1[gpu_fieldn_index(x, y, z, 22)] = h2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
	h1[gpu_fieldn_index(x, y, z, 23)] = h2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
	h1[gpu_fieldn_index(x, y, z, 24)] = h2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
	h1[gpu_fieldn_index(x, y, z, 25)] = h2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
	h1[gpu_fieldn_index(x, y, z, 26)] = h2[gpu_fieldn_index(xm1, yp1, zp1, 26)];

	// negative charges
	hn1[gpu_fieldn_index(x, y, z, 1)] = hn2[gpu_fieldn_index(xm1, y, z, 1)];
	hn1[gpu_fieldn_index(x, y, z, 2)] = hn2[gpu_fieldn_index(xp1, y, z, 2)];
	hn1[gpu_fieldn_index(x, y, z, 3)] = hn2[gpu_fieldn_index(x, ym1, z, 3)];
	hn1[gpu_fieldn_index(x, y, z, 4)] = hn2[gpu_fieldn_index(x, yp1, z, 4)];
	hn1[gpu_fieldn_index(x, y, z, 5)] = hn2[gpu_fieldn_index(x, y, zm1, 5)];
	hn1[gpu_fieldn_index(x, y, z, 6)] = hn2[gpu_fieldn_index(x, y, zp1, 6)];
	hn1[gpu_fieldn_index(x, y, z, 7)] = hn2[gpu_fieldn_index(xm1, ym1, z, 7)];
	hn1[gpu_fieldn_index(x, y, z, 8)] = hn2[gpu_fieldn_index(xp1, yp1, z, 8)];
	hn1[gpu_fieldn_index(x, y, z, 9)] = hn2[gpu_fieldn_index(xm1, y, zm1, 9)];
	hn1[gpu_fieldn_index(x, y, z, 10)] = hn2[gpu_fieldn_index(xp1, y, zp1, 10)];
	hn1[gpu_fieldn_index(x, y, z, 11)] = hn2[gpu_fieldn_index(x, ym1, zm1, 11)];
	hn1[gpu_fieldn_index(x, y, z, 12)] = hn2[gpu_fieldn_index(x, yp1, zp1, 12)];
	hn1[gpu_fieldn_index(x, y, z, 13)] = hn2[gpu_fieldn_index(xm1, yp1, z, 13)];
	hn1[gpu_fieldn_index(x, y, z, 14)] = hn2[gpu_fieldn_index(xp1, ym1, z, 14)];
	hn1[gpu_fieldn_index(x, y, z, 15)] = hn2[gpu_fieldn_index(xm1, y, zp1, 15)];
	hn1[gpu_fieldn_index(x, y, z, 16)] = hn2[gpu_fieldn_index(xp1, y, zm1, 16)];
	hn1[gpu_fieldn_index(x, y, z, 17)] = hn2[gpu_fieldn_index(x, ym1, zp1, 17)];
	hn1[gpu_fieldn_index(x, y, z, 18)] = hn2[gpu_fieldn_index(x, yp1, zm1, 18)];
	hn1[gpu_fieldn_index(x, y, z, 19)] = hn2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
	hn1[gpu_fieldn_index(x, y, z, 20)] = hn2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
	hn1[gpu_fieldn_index(x, y, z, 21)] = hn2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
	hn1[gpu_fieldn_index(x, y, z, 22)] = hn2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
	hn1[gpu_fieldn_index(x, y, z, 23)] = hn2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
	hn1[gpu_fieldn_index(x, y, z, 24)] = hn2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
	hn1[gpu_fieldn_index(x, y, z, 25)] = hn2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
	hn1[gpu_fieldn_index(x, y, z, 26)] = hn2[gpu_fieldn_index(xm1, yp1, zp1, 26)];

	// temperature
	temp1[gpu_fieldn_index(x, y, z, 1)] = temp2[gpu_fieldn_index(xm1, y, z, 1)];
	temp1[gpu_fieldn_index(x, y, z, 2)] = temp2[gpu_fieldn_index(xp1, y, z, 2)];
	temp1[gpu_fieldn_index(x, y, z, 3)] = temp2[gpu_fieldn_index(x, ym1, z, 3)];
	temp1[gpu_fieldn_index(x, y, z, 4)] = temp2[gpu_fieldn_index(x, yp1, z, 4)];
	temp1[gpu_fieldn_index(x, y, z, 5)] = temp2[gpu_fieldn_index(x, y, zm1, 5)];
	temp1[gpu_fieldn_index(x, y, z, 6)] = temp2[gpu_fieldn_index(x, y, zp1, 6)];
	temp1[gpu_fieldn_index(x, y, z, 7)] = temp2[gpu_fieldn_index(xm1, ym1, z, 7)];
	temp1[gpu_fieldn_index(x, y, z, 8)] = temp2[gpu_fieldn_index(xp1, yp1, z, 8)];
	temp1[gpu_fieldn_index(x, y, z, 9)] = temp2[gpu_fieldn_index(xm1, y, zm1, 9)];
	temp1[gpu_fieldn_index(x, y, z, 10)] = temp2[gpu_fieldn_index(xp1, y, zp1, 10)];
	temp1[gpu_fieldn_index(x, y, z, 11)] = temp2[gpu_fieldn_index(x, ym1, zm1, 11)];
	temp1[gpu_fieldn_index(x, y, z, 12)] = temp2[gpu_fieldn_index(x, yp1, zp1, 12)];
	temp1[gpu_fieldn_index(x, y, z, 13)] = temp2[gpu_fieldn_index(xm1, yp1, z, 13)];
	temp1[gpu_fieldn_index(x, y, z, 14)] = temp2[gpu_fieldn_index(xp1, ym1, z, 14)];
	temp1[gpu_fieldn_index(x, y, z, 15)] = temp2[gpu_fieldn_index(xm1, y, zp1, 15)];
	temp1[gpu_fieldn_index(x, y, z, 16)] = temp2[gpu_fieldn_index(xp1, y, zm1, 16)];
	temp1[gpu_fieldn_index(x, y, z, 17)] = temp2[gpu_fieldn_index(x, ym1, zp1, 17)];
	temp1[gpu_fieldn_index(x, y, z, 18)] = temp2[gpu_fieldn_index(x, yp1, zm1, 18)];
	temp1[gpu_fieldn_index(x, y, z, 19)] = temp2[gpu_fieldn_index(xm1, ym1, zm1, 19)];
	temp1[gpu_fieldn_index(x, y, z, 20)] = temp2[gpu_fieldn_index(xp1, yp1, zp1, 20)];
	temp1[gpu_fieldn_index(x, y, z, 21)] = temp2[gpu_fieldn_index(xm1, ym1, zp1, 21)];
	temp1[gpu_fieldn_index(x, y, z, 22)] = temp2[gpu_fieldn_index(xp1, yp1, zm1, 22)];
	temp1[gpu_fieldn_index(x, y, z, 23)] = temp2[gpu_fieldn_index(xm1, yp1, zm1, 23)];
	temp1[gpu_fieldn_index(x, y, z, 24)] = temp2[gpu_fieldn_index(xp1, ym1, zp1, 24)];
	temp1[gpu_fieldn_index(x, y, z, 25)] = temp2[gpu_fieldn_index(xp1, ym1, zm1, 25)];
	temp1[gpu_fieldn_index(x, y, z, 26)] = temp2[gpu_fieldn_index(xm1, yp1, zp1, 26)];
}

__global__ void gpu_bc_charge(double *h0, double *h1, double *h2, double *hn0, double *hn1, double *hn2, double *temp0, double *temp1, double *temp2)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y;
	unsigned int z = blockIdx.z;
	
	// No-flux boundary conditions as in Yoshida - 2014 - Coupled lattice Boltzmann method for simulating electrokinetic flows - a localized scheme for the Nernst-Planck model
	if (z == 0 || z == NZ - 1) {	
		// positive charge
		double ht1 = h2[gpu_fieldn_index(x, y, z, 1)];
		double ht2 = h2[gpu_fieldn_index(x, y, z, 2)];
		double ht3 = h2[gpu_fieldn_index(x, y, z, 3)];
		double ht4 = h2[gpu_fieldn_index(x, y, z, 4)];
		double ht5 = h2[gpu_fieldn_index(x, y, z, 5)];
		double ht6 = h2[gpu_fieldn_index(x, y, z, 6)];
		double ht7 = h2[gpu_fieldn_index(x, y, z, 7)];
		double ht8 = h2[gpu_fieldn_index(x, y, z, 8)];
		double ht9 = h2[gpu_fieldn_index(x, y, z, 9)];
		double ht10 = h2[gpu_fieldn_index(x, y, z, 10)];
		double ht11 = h2[gpu_fieldn_index(x, y, z, 11)];
		double ht12 = h2[gpu_fieldn_index(x, y, z, 12)];
		double ht13 = h2[gpu_fieldn_index(x, y, z, 13)];
		double ht14 = h2[gpu_fieldn_index(x, y, z, 14)];
		double ht15 = h2[gpu_fieldn_index(x, y, z, 15)];
		double ht16 = h2[gpu_fieldn_index(x, y, z, 16)];
		double ht17 = h2[gpu_fieldn_index(x, y, z, 17)];
		double ht18 = h2[gpu_fieldn_index(x, y, z, 18)];
		double ht19 = h2[gpu_fieldn_index(x, y, z, 19)];
		double ht20 = h2[gpu_fieldn_index(x, y, z, 20)];
		double ht21 = h2[gpu_fieldn_index(x, y, z, 21)];
		double ht22 = h2[gpu_fieldn_index(x, y, z, 22)];
		double ht23 = h2[gpu_fieldn_index(x, y, z, 23)];
		double ht24 = h2[gpu_fieldn_index(x, y, z, 24)];
		double ht25 = h2[gpu_fieldn_index(x, y, z, 25)];
		double ht26 = h2[gpu_fieldn_index(x, y, z, 26)];

		h0[gpu_field0_index(x, y, z)] = h0[gpu_field0_index(x, y, z)];
		h1[gpu_fieldn_index(x, y, z, 1)] = ht2;
		h1[gpu_fieldn_index(x, y, z, 2)] = ht1;
		h1[gpu_fieldn_index(x, y, z, 3)] = ht4;
		h1[gpu_fieldn_index(x, y, z, 4)] = ht3;
		h1[gpu_fieldn_index(x, y, z, 5)] = ht6;
		h1[gpu_fieldn_index(x, y, z, 6)] = ht5;

		h1[gpu_fieldn_index(x, y, z, 7)] = ht8;
		h1[gpu_fieldn_index(x, y, z, 8)] = ht7;
		h1[gpu_fieldn_index(x, y, z, 9)] = ht10;
		h1[gpu_fieldn_index(x, y, z, 10)] = ht9;
		h1[gpu_fieldn_index(x, y, z, 11)] = ht12;
		h1[gpu_fieldn_index(x, y, z, 12)] = ht11;
		h1[gpu_fieldn_index(x, y, z, 13)] = ht14;
		h1[gpu_fieldn_index(x, y, z, 14)] = ht13;
		h1[gpu_fieldn_index(x, y, z, 15)] = ht16;
		h1[gpu_fieldn_index(x, y, z, 16)] = ht15;
		h1[gpu_fieldn_index(x, y, z, 17)] = ht18;
		h1[gpu_fieldn_index(x, y, z, 18)] = ht17;

		h1[gpu_fieldn_index(x, y, z, 19)] = ht20;
		h1[gpu_fieldn_index(x, y, z, 20)] = ht19;
		h1[gpu_fieldn_index(x, y, z, 21)] = ht22;
		h1[gpu_fieldn_index(x, y, z, 22)] = ht21;
		h1[gpu_fieldn_index(x, y, z, 23)] = ht24;
		h1[gpu_fieldn_index(x, y, z, 24)] = ht23;
		h1[gpu_fieldn_index(x, y, z, 25)] = ht26;
		h1[gpu_fieldn_index(x, y, z, 26)] = ht25;

		// negative charge
		ht1 = hn2[gpu_fieldn_index(x, y, z, 1)];
		ht2 = hn2[gpu_fieldn_index(x, y, z, 2)];
		ht3 = hn2[gpu_fieldn_index(x, y, z, 3)];
		ht4 = hn2[gpu_fieldn_index(x, y, z, 4)];
		ht5 = hn2[gpu_fieldn_index(x, y, z, 5)];
		ht6 = hn2[gpu_fieldn_index(x, y, z, 6)];
		ht7 = hn2[gpu_fieldn_index(x, y, z, 7)];
		ht8 = hn2[gpu_fieldn_index(x, y, z, 8)];
		ht9 = hn2[gpu_fieldn_index(x, y, z, 9)];
		ht10 = hn2[gpu_fieldn_index(x, y, z, 10)];
		ht11 = hn2[gpu_fieldn_index(x, y, z, 11)];
		ht12 = hn2[gpu_fieldn_index(x, y, z, 12)];
		ht13 = hn2[gpu_fieldn_index(x, y, z, 13)];
		ht14 = hn2[gpu_fieldn_index(x, y, z, 14)];
		ht15 = hn2[gpu_fieldn_index(x, y, z, 15)];
		ht16 = hn2[gpu_fieldn_index(x, y, z, 16)];
		ht17 = hn2[gpu_fieldn_index(x, y, z, 17)];
		ht18 = hn2[gpu_fieldn_index(x, y, z, 18)];
		ht19 = hn2[gpu_fieldn_index(x, y, z, 19)];
		ht20 = hn2[gpu_fieldn_index(x, y, z, 20)];
		ht21 = hn2[gpu_fieldn_index(x, y, z, 21)];
		ht22 = hn2[gpu_fieldn_index(x, y, z, 22)];
		ht23 = hn2[gpu_fieldn_index(x, y, z, 23)];
		ht24 = hn2[gpu_fieldn_index(x, y, z, 24)];
		ht25 = hn2[gpu_fieldn_index(x, y, z, 25)];
		ht26 = hn2[gpu_fieldn_index(x, y, z, 26)];

		hn0[gpu_field0_index(x, y, z)] = hn0[gpu_field0_index(x, y, z)];
		hn1[gpu_fieldn_index(x, y, z, 1)] = ht2;
		hn1[gpu_fieldn_index(x, y, z, 2)] = ht1;
		hn1[gpu_fieldn_index(x, y, z, 3)] = ht4;
		hn1[gpu_fieldn_index(x, y, z, 4)] = ht3;
		hn1[gpu_fieldn_index(x, y, z, 5)] = ht6;
		hn1[gpu_fieldn_index(x, y, z, 6)] = ht5;

		hn1[gpu_fieldn_index(x, y, z, 7)] = ht8;
		hn1[gpu_fieldn_index(x, y, z, 8)] = ht7;
		hn1[gpu_fieldn_index(x, y, z, 9)] = ht10;
		hn1[gpu_fieldn_index(x, y, z, 10)] = ht9;
		hn1[gpu_fieldn_index(x, y, z, 11)] = ht12;
		hn1[gpu_fieldn_index(x, y, z, 12)] = ht11;
		hn1[gpu_fieldn_index(x, y, z, 13)] = ht14;
		hn1[gpu_fieldn_index(x, y, z, 14)] = ht13;
		hn1[gpu_fieldn_index(x, y, z, 15)] = ht16;
		hn1[gpu_fieldn_index(x, y, z, 16)] = ht15;
		hn1[gpu_fieldn_index(x, y, z, 17)] = ht18;
		hn1[gpu_fieldn_index(x, y, z, 18)] = ht17;

		hn1[gpu_fieldn_index(x, y, z, 19)] = ht20;
		hn1[gpu_fieldn_index(x, y, z, 20)] = ht19;
		hn1[gpu_fieldn_index(x, y, z, 21)] = ht22;
		hn1[gpu_fieldn_index(x, y, z, 22)] = ht21;
		hn1[gpu_fieldn_index(x, y, z, 23)] = ht24;
		hn1[gpu_fieldn_index(x, y, z, 24)] = ht23;
		hn1[gpu_fieldn_index(x, y, z, 25)] = ht26;
		hn1[gpu_fieldn_index(x, y, z, 26)] = ht25;
	}

	if (z == 0) {
		//double multi0c = 2.0*charge0*w0;
		//double multisc = 2.0*charge0*ws;
		//double multiac = 2.0*charge0*wa;
		//double multidc = 2.0*charge0*wd;

		double multi0T = 2.0*TH*w0;
		double multisT = 2.0*TH*ws;
		double multiaT = 2.0*TH*wa;
		double multidT = 2.0*TH*wd;

		/*
		// lower plate for charge density
		double ht1 = h2[gpu_fieldn_index(x, y, 0, 1)];
		double ht2 = h2[gpu_fieldn_index(x, y, 0, 2)];
		double ht3 = h2[gpu_fieldn_index(x, y, 0, 3)];
		double ht4 = h2[gpu_fieldn_index(x, y, 0, 4)];
		double ht5 = h2[gpu_fieldn_index(x, y, 0, 5)];
		double ht6 = h2[gpu_fieldn_index(x, y, 0, 6)];
		double ht7 = h2[gpu_fieldn_index(x, y, 0, 7)];
		double ht8 = h2[gpu_fieldn_index(x, y, 0, 8)];
		double ht9 = h2[gpu_fieldn_index(x, y, 0, 9)];
		double ht10 = h2[gpu_fieldn_index(x, y, 0, 10)];
		double ht11 = h2[gpu_fieldn_index(x, y, 0, 11)];
		double ht12 = h2[gpu_fieldn_index(x, y, 0, 12)];
		double ht13 = h2[gpu_fieldn_index(x, y, 0, 13)];
		double ht14 = h2[gpu_fieldn_index(x, y, 0, 14)];
		double ht15 = h2[gpu_fieldn_index(x, y, 0, 15)];
		double ht16 = h2[gpu_fieldn_index(x, y, 0, 16)];
		double ht17 = h2[gpu_fieldn_index(x, y, 0, 17)];
		double ht18 = h2[gpu_fieldn_index(x, y, 0, 18)];
		double ht19 = h2[gpu_fieldn_index(x, y, 0, 19)];
		double ht20 = h2[gpu_fieldn_index(x, y, 0, 20)];
		double ht21 = h2[gpu_fieldn_index(x, y, 0, 21)];
		double ht22 = h2[gpu_fieldn_index(x, y, 0, 22)];
		double ht23 = h2[gpu_fieldn_index(x, y, 0, 23)];
		double ht24 = h2[gpu_fieldn_index(x, y, 0, 24)];
		double ht25 = h2[gpu_fieldn_index(x, y, 0, 25)];
		double ht26 = h2[gpu_fieldn_index(x, y, 0, 26)];

		h0[gpu_field0_index(x, y, 0)] = -h0[gpu_field0_index(x, y, 0)] + multi0c;
		h1[gpu_fieldn_index(x, y, 0, 1)] = -ht2 + multisc;
		h1[gpu_fieldn_index(x, y, 0, 2)] = -ht1 + multisc;
		h1[gpu_fieldn_index(x, y, 0, 3)] = -ht4 + multisc;
		h1[gpu_fieldn_index(x, y, 0, 4)] = -ht3 + multisc;
		h1[gpu_fieldn_index(x, y, 0, 5)] = -ht6 + multisc;
		h1[gpu_fieldn_index(x, y, 0, 6)] = -ht5 + multisc;

		h1[gpu_fieldn_index(x, y, 0, 7)] = -ht8 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 8)] = -ht7 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 9)] = -ht10 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 10)] = -ht9 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 11)] = -ht12 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 12)] = -ht11 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 13)] = -ht14 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 14)] = -ht13 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 15)] = -ht16 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 16)] = -ht15 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 17)] = -ht18 + multiac;
		h1[gpu_fieldn_index(x, y, 0, 18)] = -ht17 + multiac;

		h1[gpu_fieldn_index(x, y, 0, 19)] = -ht20 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 20)] = -ht19 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 21)] = -ht22 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 22)] = -ht21 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 23)] = -ht24 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 24)] = -ht23 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 25)] = -ht26 + multidc;
		h1[gpu_fieldn_index(x, y, 0, 26)] = -ht25 + multidc;
		//======================================================================================================================================================================================
		//if (x == 0 && y == 0 && z == 0) test = h1[gpu_fieldn_index(x, y, z, 21)];// h1[gpu_fieldn_index(x, y, z, 1)];
		//======================================================================================================================================================================================
		*/
		// lower plate for temperature
		double tempt1 = temp2[gpu_fieldn_index(x, y, 0, 1)];
		double tempt2 = temp2[gpu_fieldn_index(x, y, 0, 2)];
		double tempt3 = temp2[gpu_fieldn_index(x, y, 0, 3)];
		double tempt4 = temp2[gpu_fieldn_index(x, y, 0, 4)];
		double tempt5 = temp2[gpu_fieldn_index(x, y, 0, 5)];
		double tempt6 = temp2[gpu_fieldn_index(x, y, 0, 6)];
		double tempt7 = temp2[gpu_fieldn_index(x, y, 0, 7)];
		double tempt8 = temp2[gpu_fieldn_index(x, y, 0, 8)];
		double tempt9 = temp2[gpu_fieldn_index(x, y, 0, 9)];
		double tempt10 = temp2[gpu_fieldn_index(x, y, 0, 10)];
		double tempt11 = temp2[gpu_fieldn_index(x, y, 0, 11)];
		double tempt12 = temp2[gpu_fieldn_index(x, y, 0, 12)];
		double tempt13 = temp2[gpu_fieldn_index(x, y, 0, 13)];
		double tempt14 = temp2[gpu_fieldn_index(x, y, 0, 14)];
		double tempt15 = temp2[gpu_fieldn_index(x, y, 0, 15)];
		double tempt16 = temp2[gpu_fieldn_index(x, y, 0, 16)];
		double tempt17 = temp2[gpu_fieldn_index(x, y, 0, 17)];
		double tempt18 = temp2[gpu_fieldn_index(x, y, 0, 18)];
		double tempt19 = temp2[gpu_fieldn_index(x, y, 0, 19)];
		double tempt20 = temp2[gpu_fieldn_index(x, y, 0, 20)];
		double tempt21 = temp2[gpu_fieldn_index(x, y, 0, 21)];
		double tempt22 = temp2[gpu_fieldn_index(x, y, 0, 22)];
		double tempt23 = temp2[gpu_fieldn_index(x, y, 0, 23)];
		double tempt24 = temp2[gpu_fieldn_index(x, y, 0, 24)];
		double tempt25 = temp2[gpu_fieldn_index(x, y, 0, 25)];
		double tempt26 = temp2[gpu_fieldn_index(x, y, 0, 26)];

		temp0[gpu_field0_index(x, y, 0)] = -temp0[gpu_field0_index(x, y, 0)] + multi0T;
		temp1[gpu_fieldn_index(x, y, 0, 1)] = -tempt2 + multisT;
		temp1[gpu_fieldn_index(x, y, 0, 2)] = -tempt1 + multisT;
		temp1[gpu_fieldn_index(x, y, 0, 3)] = -tempt4 + multisT;
		temp1[gpu_fieldn_index(x, y, 0, 4)] = -tempt3 + multisT;
		temp1[gpu_fieldn_index(x, y, 0, 5)] = -tempt6 + multisT;
		temp1[gpu_fieldn_index(x, y, 0, 6)] = -tempt5 + multisT;

		temp1[gpu_fieldn_index(x, y, 0, 7)] = -tempt8 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 8)] = -tempt7 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 9)] = -tempt10 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 10)] = -tempt9 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 11)] = -tempt12 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 12)] = -tempt11 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 13)] = -tempt14 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 14)] = -tempt13 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 15)] = -tempt16 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 16)] = -tempt15 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 17)] = -tempt18 + multiaT;
		temp1[gpu_fieldn_index(x, y, 0, 18)] = -tempt17 + multiaT;

		temp1[gpu_fieldn_index(x, y, 0, 19)] = -tempt20 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 20)] = -tempt19 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 21)] = -tempt22 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 22)] = -tempt21 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 23)] = -tempt24 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 24)] = -tempt23 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 25)] = -tempt26 + multidT;
		temp1[gpu_fieldn_index(x, y, 0, 26)] = -tempt25 + multidT;	
	}


	
	if (z == NZ - 1) {
		
		// upper plate zero temperature
		double tempt1 = temp2[gpu_fieldn_index(x, y, z, 1)];
		double tempt2 = temp2[gpu_fieldn_index(x, y, z, 2)];
		double tempt3 = temp2[gpu_fieldn_index(x, y, z, 3)];
		double tempt4 = temp2[gpu_fieldn_index(x, y, z, 4)];
		double tempt5 = temp2[gpu_fieldn_index(x, y, z, 5)];
		double tempt6 = temp2[gpu_fieldn_index(x, y, z, 6)];
		double tempt7 = temp2[gpu_fieldn_index(x, y, z, 7)];
		double tempt8 = temp2[gpu_fieldn_index(x, y, z, 8)];
		double tempt9 = temp2[gpu_fieldn_index(x, y, z, 9)];
		double tempt10 = temp2[gpu_fieldn_index(x, y, z, 10)];
		double tempt11 = temp2[gpu_fieldn_index(x, y, z, 11)];
		double tempt12 = temp2[gpu_fieldn_index(x, y, z, 12)];
		double tempt13 = temp2[gpu_fieldn_index(x, y, z, 13)];
		double tempt14 = temp2[gpu_fieldn_index(x, y, z, 14)];
		double tempt15 = temp2[gpu_fieldn_index(x, y, z, 15)];
		double tempt16 = temp2[gpu_fieldn_index(x, y, z, 16)];
		double tempt17 = temp2[gpu_fieldn_index(x, y, z, 17)];
		double tempt18 = temp2[gpu_fieldn_index(x, y, z, 18)];
		double tempt19 = temp2[gpu_fieldn_index(x, y, z, 19)];
		double tempt20 = temp2[gpu_fieldn_index(x, y, z, 20)];
		double tempt21 = temp2[gpu_fieldn_index(x, y, z, 21)];
		double tempt22 = temp2[gpu_fieldn_index(x, y, z, 22)];
		double tempt23 = temp2[gpu_fieldn_index(x, y, z, 23)];
		double tempt24 = temp2[gpu_fieldn_index(x, y, z, 24)];
		double tempt25 = temp2[gpu_fieldn_index(x, y, z, 25)];
		double tempt26 = temp2[gpu_fieldn_index(x, y, z, 26)];

		temp0[gpu_field0_index(x, y, z)] = -temp0[gpu_field0_index(x, y, z)];
		temp1[gpu_fieldn_index(x, y, z, 1)] = -tempt2;
		temp1[gpu_fieldn_index(x, y, z, 2)] = -tempt1;
		temp1[gpu_fieldn_index(x, y, z, 3)] = -tempt4;
		temp1[gpu_fieldn_index(x, y, z, 4)] = -tempt3;
		temp1[gpu_fieldn_index(x, y, z, 5)] = -tempt6;
		temp1[gpu_fieldn_index(x, y, z, 6)] = -tempt5;

		temp1[gpu_fieldn_index(x, y, z, 7)] = -tempt8;
		temp1[gpu_fieldn_index(x, y, z, 8)] = -tempt7;
		temp1[gpu_fieldn_index(x, y, z, 9)] = -tempt10;
		temp1[gpu_fieldn_index(x, y, z, 10)] = -tempt9;
		temp1[gpu_fieldn_index(x, y, z, 11)] = -tempt12;
		temp1[gpu_fieldn_index(x, y, z, 12)] = -tempt11;
		temp1[gpu_fieldn_index(x, y, z, 13)] = -tempt14;
		temp1[gpu_fieldn_index(x, y, z, 14)] = -tempt13;
		temp1[gpu_fieldn_index(x, y, z, 15)] = -tempt16;
		temp1[gpu_fieldn_index(x, y, z, 16)] = -tempt15;
		temp1[gpu_fieldn_index(x, y, z, 17)] = -tempt18;
		temp1[gpu_fieldn_index(x, y, z, 18)] = -tempt17;

		temp1[gpu_fieldn_index(x, y, z, 19)] = -tempt20;
		temp1[gpu_fieldn_index(x, y, z, 20)] = -tempt19;
		temp1[gpu_fieldn_index(x, y, z, 21)] = -tempt22;
		temp1[gpu_fieldn_index(x, y, z, 22)] = -tempt21;
		temp1[gpu_fieldn_index(x, y, z, 23)] = -tempt24;
		temp1[gpu_fieldn_index(x, y, z, 24)] = -tempt23;
		temp1[gpu_fieldn_index(x, y, z, 25)] = -tempt26;
		temp1[gpu_fieldn_index(x, y, z, 26)] = -tempt25;
	}


}


__host__ void compute_parameters(double *T, double *M, double *C, double *Fe, double *Pr) {
	double K_host;
	double eps_host;
	double voltage_host;
	//double nu_host;
	double Ly_host;
	double diffu_host;
	double charge0_host;
	double rho0_host;
	double D_host;

	hipMemcpyFromSymbol(&K_host, HIP_SYMBOL(K), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&eps_host, HIP_SYMBOL(eps), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&voltage_host, HIP_SYMBOL(voltage), sizeof(double), 0, hipMemcpyDeviceToHost);
	//hipMemcpyFromSymbol(&nu_host, HIP_SYMBOL(nu), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Lz_host, HIP_SYMBOL(Lz), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&diffu_host, HIP_SYMBOL(diffu), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&charge0_host, HIP_SYMBOL(chargeinf), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&rho0_host, HIP_SYMBOL(rho0), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&D_host, HIP_SYMBOL(D), sizeof(double), 0, hipMemcpyDeviceToHost);

	*M = sqrt(eps_host / rho0_host) / K_host;
	*T = eps_host*voltage_host / K_host / nu_host / rho0_host;
	*C = charge0_host * Lz_host * Lz_host / (voltage_host * eps_host);
	*Fe = K_host * voltage_host / diffu_host;
	*Pr = nu_host / D_host;

}

__host__ void report_flow_properties(unsigned int n, double t, double *rho, 
	double *charge, double *phi, double *ux, double *uy, double *uz, double *Ex, double *Ey, double *Ez)
{
    printf("Iteration: %u, physical time: %g.\n",n,t);
}

__host__ void save_scalar(const char* name, double *scalar_gpu, double *scalar_host, unsigned int n)
{
    // assume reasonably-sized file names
    char filename[128];
    char format[16];
    
    // compute maximum number of digits
    int ndigits = floor(log10((double)NSTEPS)+1.0);
    
    // generate format string
    // file name format is name0000nnn.bin
    sprintf(format,"%%s%%0%dd.bin",ndigits);
    sprintf(filename,format,name,n);
    
    // transfer memory from GPU to host
    checkCudaErrors(hipMemcpy(scalar_host,scalar_gpu,mem_size_scalar,hipMemcpyDeviceToHost));
    
    // open file for writing
    FILE *fout = fopen(filename,"wb+");
    
    // write data
    fwrite(scalar_host,1,mem_size_scalar,fout);
    
    // close file
    fclose(fout);
    
    if(ferror(fout))
    {
        fprintf(stderr,"Error saving to %s\n",filename);
        perror("");
    }
    else
    {
        if(!quiet)
            printf("Saved to %s\n",filename);
    }
}

__host__
void save_data_tecplot(FILE *fout, double time, double *rho_gpu, double *charge_gpu, double *chargen_gpu, double *phi_gpu,
	double *ux_gpu, double *uy_gpu, double *uz_gpu, double *Ex_gpu, double *Ey_gpu, double *Ez_gpu, double *Temp_gpu, int first) {
	
	double *rho    = (double*)malloc(mem_size_scalar);
	double *charge = (double*)malloc(mem_size_scalar);
	double *chargen = (double*)malloc(mem_size_scalar);
	double *phi    = (double*)malloc(mem_size_scalar);
	double *Temp = (double*)malloc(mem_size_scalar);
	double *ux     = (double*)malloc(mem_size_scalar);
	double *uy     = (double*)malloc(mem_size_scalar);
	double *uz     = (double*)malloc(mem_size_scalar);
	double *Ex     = (double*)malloc(mem_size_scalar);
	double *Ey     = (double*)malloc(mem_size_scalar);
	double *Ez     = (double*)malloc(mem_size_scalar);
	double dx_host;
	double dy_host;
	double dz_host;
	// transfer memory from GPU to host
	checkCudaErrors(hipMemcpy(rho,    rho_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(charge, charge_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(chargen, chargen_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(phi,    phi_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Temp,   Temp_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(ux,     ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uy,     uy_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uz,     uz_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ex,     Ex_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ey,     Ey_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ez,     Ez_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	hipMemcpyFromSymbol(&dx_host, HIP_SYMBOL(dx), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dy_host, HIP_SYMBOL(dy), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dz_host, HIP_SYMBOL(dz), sizeof(double), 0, hipMemcpyDeviceToHost);	
	
	// apply boundary conditions (upper and lower plate)
	for (unsigned int y = 0; y < NY; ++y) {
		for (unsigned int x = 0; x < NX; ++x) {
			rho[scalar_index(x, y, 0)] = 2.0*rho[scalar_index(x, y, 1)] - rho[scalar_index(x, y, 2)];
			charge[scalar_index(x, y, 0)] = 2.0*charge[scalar_index(x, y, 1)] - charge[scalar_index(x, y, 2)];
			chargen[scalar_index(x, y, 0)] = 2.0*chargen[scalar_index(x, y, 1)] - chargen[scalar_index(x, y, 2)];
			ux[scalar_index(x, y, 0)] = 2.0*ux[scalar_index(x, y, 1)] - ux[scalar_index(x, y, 2)];
			uy[scalar_index(x, y, 0)] = 2.0*uy[scalar_index(x, y, 1)] - uy[scalar_index(x, y, 2)];
			uz[scalar_index(x, y, 0)] = 2.0*uz[scalar_index(x, y, 1)] - uz[scalar_index(x, y, 2)];
			rho[scalar_index(x, y, NZ - 1)] = 2.0*rho[scalar_index(x, y, NZ - 2)] - rho[scalar_index(x, y, NZ - 3)];
			charge[scalar_index(x, y, NZ - 1)] = 2.0*charge[scalar_index(x, y, NZ - 2)] - charge[scalar_index(x, y, NZ - 3)];
			chargen[scalar_index(x, y, NZ - 1)] = 2.0*chargen[scalar_index(x, y, NZ - 2)] - chargen[scalar_index(x, y, NZ - 3)];
			ux[scalar_index(x, y, NZ - 1)] = 2.0*ux[scalar_index(x, y, NZ - 2)] - ux[scalar_index(x, y, NZ - 3)];
			uy[scalar_index(x, y, NZ - 1)] = 2.0*uy[scalar_index(x, y, NZ - 2)] - uy[scalar_index(x, y, NZ - 3)];
			uz[scalar_index(x, y, NZ - 1)] = 2.0*uz[scalar_index(x, y, NZ - 2)] - uz[scalar_index(x, y, NZ - 3)];
		}
	}

	if (first)
	{
		char str[] = "VARIABLES=\"x\",\"y\",\"z\",\"u\",\"v\",\"w\",\"p\",\"charge\",\"neg charge\",\"phi\",\"Ex\",\"Ey\",\"Ez\",\"Temperature\"";
		//fwrite(str, 1, sizeof(str), fout);
		fprintf(fout, "%s\n", str);
	}
	fprintf(fout, "\n");
	fprintf(fout, "ZONE T=\"t=%g\", F=POINT, I = %d, J = %d, K = %d\n", time, NX, NY, NZ);

	for (unsigned int z = 0; z < NZ; ++z) 
	{
		for (unsigned int y = 0; y < NY; ++y)
		{
			for (unsigned int x = 0; x < NX; ++x)
			{
				fprintf(fout, "%g %g %g %g %g %g %g %g %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f\n", dx_host*x, dy_host*y, dz_host*z,
					ux[scalar_index(x, y, z)], uy[scalar_index(x, y, z)], uz[scalar_index(x, y, z)], rho[scalar_index(x, y, z)], charge[scalar_index(x, y, z)], chargen[scalar_index(x, y, z)],
					phi[scalar_index(x, y, z)], Ex[scalar_index(x, y, z)], Ey[scalar_index(x, y, z)], Ez[scalar_index(x, y, z)], Temp[scalar_index(x, y, z)]);
			}
		}
	}
}

__host__
void save_data_end(FILE *fend, double time, double *rho_gpu, double *charge_gpu, double *chargen_gpu, double *phi_gpu,
	double *ux_gpu, double *uy_gpu, double *uz_gpu, double *Ex_gpu, double *Ey_gpu, double *Ez_gpu, double *Temp_gpu) {

	double *rho = (double*)malloc(mem_size_scalar);
	double *charge = (double*)malloc(mem_size_scalar);
	double *chargen = (double*)malloc(mem_size_scalar);
	double *phi = (double*)malloc(mem_size_scalar);
	double *temp = (double*)malloc(mem_size_scalar);
	double *ux = (double*)malloc(mem_size_scalar);
	double *uy = (double*)malloc(mem_size_scalar);
	double *uz = (double*)malloc(mem_size_scalar);
	double *Ex = (double*)malloc(mem_size_scalar);
	double *Ey = (double*)malloc(mem_size_scalar);
	double *Ez = (double*)malloc(mem_size_scalar);

	// transfer memory from GPU to host
	checkCudaErrors(hipMemcpy(rho, rho_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(charge, charge_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(chargen, chargen_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(phi, phi_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(temp, Temp_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(ux, ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uy, uy_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uz, uz_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ex, Ex_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ey, Ey_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(Ez, Ez_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

	// apply boundary conditions (upper and lower plate)
	for (unsigned int y = 0; y < NY; ++y) {
		for (unsigned int x = 0; x < NX; ++x) {
			rho[scalar_index(x, y, 0)] = 2.0*rho[scalar_index(x, y, 1)] - rho[scalar_index(x, y, 2)];
			charge[scalar_index(x, y, 0)] = 2.0*charge[scalar_index(x, y, 1)] - charge[scalar_index(x, y, 2)];
			chargen[scalar_index(x, y, 0)] = 2.0*chargen[scalar_index(x, y, 1)] - chargen[scalar_index(x, y, 2)];
			ux[scalar_index(x, y, 0)] = 2.0*ux[scalar_index(x, y, 1)] - ux[scalar_index(x, y, 2)];
			uy[scalar_index(x, y, 0)] = 2.0*uy[scalar_index(x, y, 1)] - uy[scalar_index(x, y, 2)];
			uz[scalar_index(x, y, 0)] = 2.0*uz[scalar_index(x, y, 1)] - uz[scalar_index(x, y, 2)];
			rho[scalar_index(x, y, NZ - 1)] = 2.0*rho[scalar_index(x, y, NZ - 2)] - rho[scalar_index(x, y, NZ - 3)];
			charge[scalar_index(x, y, NZ - 1)] = 2.0*charge[scalar_index(x, y, NZ - 2)] - charge[scalar_index(x, y, NZ - 3)];
			chargen[scalar_index(x, y, NZ - 1)] = 2.0*chargen[scalar_index(x, y, NZ - 2)] - chargen[scalar_index(x, y, NZ - 3)];
			ux[scalar_index(x, y, NZ - 1)] = 2.0*ux[scalar_index(x, y, NZ - 2)] - ux[scalar_index(x, y, NZ - 3)];
			uy[scalar_index(x, y, NZ - 1)] = 2.0*uy[scalar_index(x, y, NZ - 2)] - uy[scalar_index(x, y, NZ - 3)];
			uz[scalar_index(x, y, NZ - 1)] = 2.0*uz[scalar_index(x, y, NZ - 2)] - uz[scalar_index(x, y, NZ - 3)];
		}
	}
	for (unsigned int z = 0; z < NZ; ++z)
	{
		for (unsigned int y = 0; y < NY; ++y)
		{
			for (unsigned int x = 0; x < NX; ++x)
			{
				fprintf(fend, "%10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f %10.6f\n", time,
					ux[scalar_index(x, y, z)], uy[scalar_index(x, y, z)], uz[scalar_index(x, y, z)], rho[scalar_index(x, y, z)], charge[scalar_index(x, y, z)], chargen[scalar_index(x, y, z)],
					phi[scalar_index(x, y, z)], Ex[scalar_index(x, y, z)], Ey[scalar_index(x, y, z)], Ez[scalar_index(x, y, z)], temp[scalar_index(x, y, z)]);
			}
		}
	}


}

__host__
void read_data(double *time, double *rho_gpu, double *charge_gpu, double *chargen_gpu, double *phi_gpu,
	double *ux_gpu, double *uy_gpu, double *uz_gpu, double *Ex_gpu, double *Ey_gpu, double *Ez_gpu, double *T_gpu) {

	double *rho = (double*)malloc(mem_size_scalar);
	double *charge = (double*)malloc(mem_size_scalar);
	double *chargen = (double*)malloc(mem_size_scalar);
	double *phi = (double*)malloc(mem_size_scalar);
	double *temp = (double*)malloc(mem_size_scalar);
	double *ux = (double*)malloc(mem_size_scalar);
	double *uy = (double*)malloc(mem_size_scalar);
	double *uz = (double*)malloc(mem_size_scalar);
	double *Ex = (double*)malloc(mem_size_scalar);
	double *Ey = (double*)malloc(mem_size_scalar);
	double *Ez = (double*)malloc(mem_size_scalar);

	FILE *fread = fopen("data_end.dat", "r");
	for (unsigned int z = 0; z < NZ; ++z)
	{
		for (unsigned int y = 0; y < NY; ++y)
		{
			for (unsigned int x = 0; x < NX; ++x)
			{
				fscanf(fread, "%lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf %lf", time,
					&ux[scalar_index(x, y, z)], &uy[scalar_index(x, y, z)], &uz[scalar_index(x, y, z)], &rho[scalar_index(x, y, z)], &charge[scalar_index(x, y, z)], &chargen[scalar_index(x, y, z)],
					&phi[scalar_index(x, y, z)], &Ex[scalar_index(x, y, z)], &Ey[scalar_index(x, y, z)], &Ez[scalar_index(x, y, z)], &temp[scalar_index(x, y, z)]);
			}
		}
	}
	// transfer memory from host to GPU
	checkCudaErrors(hipMemcpy(rho_gpu, rho, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(charge_gpu, charge, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(chargen_gpu, chargen, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(phi_gpu, phi, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(T_gpu, temp, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(ux_gpu, ux, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(uy_gpu, uy, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(uz_gpu, uz, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Ex_gpu, Ex, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Ey_gpu, Ey, mem_size_scalar, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Ez_gpu, Ez, mem_size_scalar, hipMemcpyHostToDevice));
	fclose(fread);
}


__host__
double current(double* c, double* cn, double* ez) {
	double I = 0;
	double K_host;

	hipMemcpyFromSymbol(&K_host, HIP_SYMBOL(K), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dz_host, HIP_SYMBOL(dz), sizeof(double), 0, hipMemcpyDeviceToHost);


	// apply boundary conditions (upper and lower plate)
	for (unsigned int z = 0; z < NZ; ++z) {
		for (unsigned int y = 0; y < NY; ++y) {
			for (unsigned int x = 0; x < NX; ++x) {
				//rho[scalar_index(x, y, 0)] = 2.0*rho[scalar_index(x, y, 1)] - rho[scalar_index(x, y, 2)];
				c[scalar_index(x, y, 0)] = 2.0*c[scalar_index(x, y, 1)] - c[scalar_index(x, y, 2)];
				cn[scalar_index(x, y, 0)] = 2.0*cn[scalar_index(x, y, 1)] - cn[scalar_index(x, y, 2)];

				//ux[scalar_index(x, y, 0)] = 2.0*ux[scalar_index(x, y, 1)] - ux[scalar_index(x, y, 2)];
				//uy[scalar_index(x, y, 0)] = 2.0*uy[scalar_index(x, y, 1)] - uy[scalar_index(x, y, 2)];
				//uz[scalar_index(x, y, 0)] = 2.0*uz[scalar_index(x, y, 1)] - uz[scalar_index(x, y, 2)];
				//rho[scalar_index(x, y, NZ - 1)] = 2.0*rho[scalar_index(x, y, NZ - 2)] - rho[scalar_index(x, y, NZ - 3)];
				c[scalar_index(x, y, NZ - 1)] = 2.0*c[scalar_index(x, y, NZ - 2)] - c[scalar_index(x, y, NZ - 3)];
				cn[scalar_index(x, y, NZ - 1)] = 2.0*cn[scalar_index(x, y, NZ - 2)] - cn[scalar_index(x, y, NZ - 3)];
				//ux[scalar_index(x, y, NZ - 1)] = 2.0*ux[scalar_index(x, y, NZ - 2)] - ux[scalar_index(x, y, NZ - 3)];
				//uy[scalar_index(x, y, NZ - 1)] = 2.0*uy[scalar_index(x, y, NZ - 2)] - uy[scalar_index(x, y, NZ - 3)];
				//uz[scalar_index(x, y, NZ - 1)] = 2.0*uz[scalar_index(x, y, NZ - 2)] - uz[scalar_index(x, y, NZ - 3)];
			}
		}
	}
	for (unsigned int y = 0; y < NY; y++) {
		for (unsigned int x = 0; x < NX; x++) {
			I += (c[scalar_index(x, y, NZ - 1)] - cn[scalar_index(x, y, NZ - 1)]) * ez[scalar_index(x, y, NZ - 1)];
		}
	}
	I = I * K_host * dz_host * dz_host;
	return I;
}

__host__
void record_umax(FILE *fend, double time, double *ux_gpu, double *uy_gpu, double *uz_gpu) {

	double *ux = (double*)malloc(mem_size_scalar);
	double *uy = (double*)malloc(mem_size_scalar);
	double *uz = (double*)malloc(mem_size_scalar);
	double umax = 0;


	// transfer memory from GPU to host
	checkCudaErrors(hipMemcpy(ux, ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uy, uy_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(uz, uz_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

	// apply boundary conditions (upper and lower plate)
	for (unsigned int y = 0; y < NY; ++y) {
		for (unsigned int x = 0; x < NX; ++x) {
			ux[scalar_index(x, y, NZ - 1)] = 2.0*ux[scalar_index(x, y, NZ - 2)] - ux[scalar_index(x, y, NZ - 3)];
			uy[scalar_index(x, y, NZ - 1)] = 2.0*uy[scalar_index(x, y, NZ - 2)] - uy[scalar_index(x, y, NZ - 3)];
			uy[scalar_index(x, y, NZ - 1)] = 2.0*uy[scalar_index(x, y, NZ - 2)] - uy[scalar_index(x, y, NZ - 3)];
		}
	}

	for (unsigned int z = 0; z < NZ; ++z)
	{
		for (unsigned int y = 0; y < NY; ++y)
		{
			for (unsigned int x = 0; x < NX; ++x)
			{
				//umax = MAX(umax, sqrt(ux[scalar_index(x, y, z)] * ux[scalar_index(x, y, z)] + uy[scalar_index(x, y, z)] * uy[scalar_index(x, y, z)]
				//	+ uz[scalar_index(x, y, z)] * uz[scalar_index(x, y, z)]));
				umax = MAX(umax, uz[scalar_index(x, y, z)]);
			}
		}
	}

	fprintf(fend, "%10.6f %10.6f\n", time, umax);

	free(ux);
	free(uy);
	free(uz);
}