#include "hip/hip_runtime.h"
/*
*   The Lattice Boltzmann Method with ETHD convection
*   Yifei Guan
*   Rice University
*   Apr/12/2020
*
*/
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>

#include "seconds.h"
#include "LBM.h"
#include "LBM.cu"
#include "poisson.cu"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

int main(int argc, char* argv[])
{
	checkCudaErrors(hipMalloc((void**)&test, sizeof(double)));

	hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Lx_host, HIP_SYMBOL(Lx), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Ly_host, HIP_SYMBOL(Ly), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dy_host, HIP_SYMBOL(dy), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Lz_host, HIP_SYMBOL(Lz), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dz_host, HIP_SYMBOL(dz), sizeof(double), 0, hipMemcpyDeviceToHost);

	hipMemcpyToSymbol(HIP_SYMBOL(nu), &nu_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(uw), &uw_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(exf), &exf_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(K), &K_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(Kn), &Kn_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(epsn), &epsn_host, sizeof(double), 0, hipMemcpyHostToDevice);

	// Compute parameters
	compute_parameters(T, M, C, Fe, Pr);

    printf("Simulating 3D electrokinetic flow with heat transfer vortices\n");
    printf("      domain size (NX x NY x NZ): %ux%ux%u\n",NX,NY,NZ);
    //printf("                T: %g\n",*T);
    //printf("                M: %g\n",*M);
    //printf("                C: %g\n",*C);
    //printf("               Fe: %g\n",*Fe);
	printf("               Ra: %g\n", Ra_host);
	printf("               Pr: %g\n", *Pr);
	printf("            uwall: %g\n",uw_host);
	printf("   External force: %g\n",exf_host);
    printf("        timesteps: %u\n",NSTEPS);
    printf("       save every: %u\n",NSAVE);
    printf("    message every: %u\n",NMSG);
    printf("\n");
    
    double bytesPerMiB = 1024.0*1024.0;
    double bytesPerGiB = 1024.0*1024.0*1024.0;
    
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));
    
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
    
    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem,&gpu_total_mem));
    
    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n",deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n",deviceProp.totalGlobalMem/bytesPerMiB);
    printf("        free memory: %.1f MiB\n",gpu_free_mem/bytesPerMiB);
    printf("\n");

	// storage of f0 at upper and lower plate
	checkCudaErrors(hipMalloc((void**)&f0bc, sizeof(double)*NX*NY*2));
    //double *prop_gpu;
	// microscopic variables
	checkCudaErrors(hipMalloc((void**)&f0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&f1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&f2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&h0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&h1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&h2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&hn0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&hn1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&hn2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&temp0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&temp1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&temp2_gpu, mem_size_n0dir));


	// macroscopic variables
	checkCudaErrors(hipMalloc((void**)&rho_gpu,    mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uy_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uz_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&charge_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&chargen_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&phi_gpu,    mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&T_gpu,		mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&Ex_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&Ey_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&Ez_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&kx,         sizeof(double)*NX));
	checkCudaErrors(hipMalloc((void**)&ky,         sizeof(double)*NY));
	checkCudaErrors(hipMalloc((void**)&kz,         sizeof(double)*NE));

    // Setup the cuFFT plan
	CHECK_CUFFT(hipfftPlan3d(&plan, NE, NY, NX, HIPFFT_Z2Z));
	//checkCudaErrors(hipMalloc((void**)&freq_gpu_ext, sizeof(hipfftDoubleComplex)*NX*NY*NE));
	//checkCudaErrors(hipMalloc((void**)&phi_gpu_ext,  sizeof(hipfftDoubleComplex)*NX*NY*NE));
	//checkCudaErrors(hipMalloc((void**)&charge_gpu_ext, sizeof(hipfftDoubleComplex)*NX*NY*NE));


	// Setup the frequencies kx and ky
	for (unsigned i = 0; i <= NX / 2; i++)
	{
		kx_host[i] = (double)i * 2.0 * M_PI / Lx_host;
	}

	for (unsigned i = NX / 2 + 1; i < NX; i++)
	{
		kx_host[i] = ((double) i - NX) * 2.0 * M_PI / Lx_host;
	}
	for (unsigned i = 0; i <= NY / 2; i++)
	{
		ky_host[i] = (double)i * 2.0 * M_PI / Ly_host;
	}

	for (unsigned i = NY / 2 + 1; i < NY; i++)
	{
		ky_host[i] = ((double)i - NY) * 2.0 * M_PI / Ly_host;
	}
	for (unsigned i = 0; i <= NE / 2; i++)
	{
		kz_host[i] = (double)i  * 2.0 * M_PI / (NE*dz_host);
	}

	for (unsigned i = NE / 2 + 1; i < NE; i++)
	{
		kz_host[i] = ((double)i - NE) * 2.0 * M_PI / (NE*dz_host);
	}

	CHECK(hipMemcpy(kx, kx_host,
		sizeof(double) * NX, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(ky, ky_host,
		sizeof(double) * NY, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(kz, kz_host,
		sizeof(double) * NE, hipMemcpyHostToDevice));
	
    // create event objects
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
	printf("Read previous data: Press 1. Start a new simulation: Press 0.\n ");
	scanf("%d", &flag);

	if (flag == 1) {
		printf("Reading previous data...\n");
		read_data(&t, rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu);
	}
	else {
		printf("Initializing...\n");
		// Zero flow at t=0
		// to initialize rho, charge, phi, ux, uy, Ex, Ey fields.
		initialization(rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu);
		t = 0;
	}

    // initialise f1,h1 as equilibrium for rho, ux, uy, charge, ex, ey
    init_equilibrium(f0_gpu,f1_gpu,h0_gpu,h1_gpu, hn0_gpu, hn1_gpu, temp0_gpu, temp1_gpu, rho_gpu,charge_gpu, chargen_gpu,
		ux_gpu,uy_gpu,uz_gpu,Ex_gpu,Ey_gpu,Ez_gpu,T_gpu);

	// open file for writing
	FILE *fout = fopen("data.dat", "wb+");
	save_data_tecplot(fout, t, rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu, 1);
	FILE *fumax = fopen("umax.dat", "wb+");

	// report computational results to screen
    //report_flow_properties(0, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu,uy_gpu, uz_gpu,Ex_gpu, Ey_gpu,Ez_gpu);
    
    double begin = seconds();
    checkCudaErrors(hipEventRecord(start,0));

    // main simulation loop; take NSTEPS time steps
	for (int i = 0; i < NSTEPS; i++) {
		// stream and collide from f1 storing to f2
		// optionally compute and save moments
		stream_collide_save(f0_gpu, f1_gpu, f2_gpu, h0_gpu, h1_gpu, h2_gpu, hn0_gpu, hn1_gpu, hn2_gpu, temp0_gpu, temp1_gpu, temp2_gpu,
			rho_gpu, charge_gpu, chargen_gpu,
			ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu, t, f0bc);
		// =========================================================================
		// Fast poisson solver
		// =========================================================================
		fast_Poisson(charge_gpu, chargen_gpu, kx, ky, kz, plan);

		t = t + dt_host;

		// =========================================================================
		// Save data for analysis
		// =========================================================================

		if (i%NSAVE == 1) {
			save_data_tecplot(fout, t, rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu, 1);
			printf("Iteration: %u, physical time: %g.\n", i, t);
		}

		if (i%printCurrent == 1) {
			checkCudaErrors(hipMemcpy(charge_host, charge_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(chargen_host, chargen_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(Ez_host, Ez_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
			double current_host = current(charge_host, chargen_host, Ez_host);
			printf("Iteration: %u, physical time: %g, Current = %g\n", i, t, current_host);
			//printf("%g\n", Ez_host[scalar_index(0, 0, 0)]);
			// =============================================================================================================
			// save umax
			// =============================================================================================================
			record_umax(fumax, t, ux_gpu, uy_gpu, uz_gpu);
		}

	}
	// end of simulation

    checkCudaErrors(hipEventRecord(stop,0));
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&milliseconds,start,stop));
    
    double end = seconds();
    double runtime = end-begin;
    double gpu_runtime = 0.001*milliseconds;	
	
	size_t doubles_read = ndir; // per node every time step
    size_t doubles_written = ndir;
    size_t doubles_saved = 3; // per node every NSAVE time steps
    
    // note NX*NY overflows when NX=NY=65536
    size_t nodes_updated = NSTEPS*size_t(NX*NY*NZ);
    size_t nodes_saved   = (NSTEPS/NSAVE)*size_t(NX*NY*NZ);
    double speed = nodes_updated/(1e6*runtime);
    
    double bandwidth = (nodes_updated*(doubles_read + doubles_written)+nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);
    
    printf(" ----- performance information -----\n");
    printf("               timesteps: %u\n",NSTEPS);
    printf("           clock runtime: %.3f (s)\n",runtime);
    printf("             gpu runtime: %.3f (s)\n",gpu_runtime);
    printf("                   speed: %.2f (Mlups)\n",speed);
    
	save_data_tecplot(fout, t, rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu, 1);
	fclose(fout);
	fclose(fumax);
	FILE *fend = fopen("data_end.dat", "wb+");
	save_data_end(fend, t, rho_gpu, charge_gpu, chargen_gpu, phi_gpu, ux_gpu, uy_gpu, uz_gpu, Ex_gpu, Ey_gpu, Ez_gpu, T_gpu);
	
    // destory event objects
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    // free all memory allocatd on the GPU and host
    checkCudaErrors(hipFree(f0_gpu));
    checkCudaErrors(hipFree(f1_gpu));
    checkCudaErrors(hipFree(f2_gpu));
	checkCudaErrors(hipFree(h0_gpu));
	checkCudaErrors(hipFree(h1_gpu));
	checkCudaErrors(hipFree(h2_gpu));
	checkCudaErrors(hipFree(hn0_gpu));
	checkCudaErrors(hipFree(hn1_gpu));
	checkCudaErrors(hipFree(hn2_gpu));
    checkCudaErrors(hipFree(rho_gpu));
	checkCudaErrors(hipFree(phi_gpu));
	checkCudaErrors(hipFree(Ex_gpu));
	checkCudaErrors(hipFree(Ey_gpu));
	checkCudaErrors(hipFree(Ez_gpu));
    checkCudaErrors(hipFree(ux_gpu));
    checkCudaErrors(hipFree(uy_gpu));
	checkCudaErrors(hipFree(uz_gpu));
	checkCudaErrors(hipFree(f0bc));
	checkCudaErrors(hipFree(kx));
	checkCudaErrors(hipFree(ky));
	checkCudaErrors(hipFree(kz));

	CHECK_CUFFT(hipfftDestroy(plan));
    //checkCudaErrors(hipFree(prop_gpu));    
	free(kx_host);
	free(ky_host);
	free(kz_host);

    // release resources associated with the GPU device
    hipDeviceReset();
	system("pause");
    return 0;
}

